#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        
        __global__ void kernNaiveScan(int n, int d, int* odata, const int* idata)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) return;

            int ipow2 = powf(2, d - 1);
            if (index >= ipow2)
            {
                odata[index] = idata[index - ipow2] + idata[index];
            }
            else
            {
                odata[index] = idata[index];
            }
        }

        __global__ void kernShiftRight(int n, int s, int* odata, const int* idata)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) return;

            int output = index >= s ? idata[index - s] : 0;
            odata[index] = output;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int *dev_odata, *dev_idata;

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc Naive::scan::dev_odata failed!");

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc Naive::scan::dev_idata failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            dim3 gridDim((n + blockSize - 1) / blockSize);

            int depth_max = ilog2ceil(n);

            timer().startGpuTimer();

            for (int d = 1; d <= depth_max; ++d)
            {
                kernNaiveScan<<<gridDim, blockSize>>>(n, d, dev_odata, dev_idata);

                int *tmp = dev_idata;
                dev_idata = dev_odata;
                dev_odata = tmp;
            }
            kernShiftRight<<<gridDim, blockSize>>>(n, 1, dev_odata, dev_idata);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_odata);
            hipFree(dev_idata);
            checkCUDAError("hipFree Naive::scan failed!");
        }
    }
}
