#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void kernScan(int n, int depth, int* odata, int* idata) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            if (idx < depth) {
                odata[idx] = idata[idx];
                return;
            }

            odata[idx] = idata[idx - depth] + idata[idx];
            return;
        }

        __global__ void kernToExclusive(int n, int* odata, int* idata) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            if (idx == 0) {
                odata[idx] = 0;
            }
            else {
                odata[idx] = idata[idx - 1];
            }
            return;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_odata;
            int* dev_idata;

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed");
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to dev_idata failed");

            timer().startGpuTimer();
            // TODO
            dim3 blocksPerGrid((n + blockSize - 1) / blockSize);

            for (int d = 1; d <= ilog2ceil(n); ++d) {
                kernScan<<<blocksPerGrid, blockSize>>>(n, 1 << (d - 1), dev_odata, dev_idata);
                std::swap(dev_odata, dev_idata);
            }
            
            // above result is inclusive, need to convert to exclusive scan
            // because of the swap, latest result is in dev_idata
            kernToExclusive<<<blocksPerGrid, blockSize>>>(n, dev_odata, dev_idata);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy from dev_odata failed");

            hipFree(dev_odata);
            checkCUDAError("hipFree dev_odata failed");
            hipFree(dev_idata);
            checkCUDAError("hipFree dev_idata failed");
        }
    }
}
