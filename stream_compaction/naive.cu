#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
		__global__ void kernNaiveScan(int n, int* odata, const int* idata, int d) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) return;

			if (index >= (1 << d)) {
				odata[index] = idata[index - (1 << d)] + idata[index];
			}
			else {
				odata[index] = idata[index];
			}
		}

		__global__ void kernShiftRight(int n, int* odata, const int* idata) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) return;

			odata[index] = index == 0 ? 0 : idata[index - 1];
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
			int blockSize = 128;

			int* dev_idata;
			int* dev_odata;
			
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");

			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy idata to dev_idata failed!");

			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
			
			for (int d = 0; d < ilog2ceil(n); d++) {
				kernNaiveScan << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, d);
				checkCUDAError("kernNaiveScan failed!");
				hipMemcpy(dev_idata, dev_odata, n * sizeof(int), hipMemcpyDeviceToDevice);
			}

			kernShiftRight << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata);
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_odata to odata failed!");


			
			hipFree(dev_idata);
			hipFree(dev_odata);

            timer().endGpuTimer();
        }
    }
}
