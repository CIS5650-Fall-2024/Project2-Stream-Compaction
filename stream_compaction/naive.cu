#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __device__ int pow_two(int d) {
            return 1 << d;
        }

        __global__ void scan_one_iteration(int n, int d, int* odata, const int* idata) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index < n) {
                int pow_d = pow_two(d - 1);
                if (index >= pow_d) {
                    odata[index] = idata[index - pow_d] + idata[index];
                }
                else {
                    odata[index] = idata[index];
                }
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* buffer1;
            int* buffer2;
            hipMalloc((void**)&buffer1, n * sizeof(int));
            checkCUDAErrorFn("failed to allocate buffer1");

            hipMalloc((void**)&buffer2, n * sizeof(int));
            checkCUDAErrorFn("failed to allocate buffer2");

            hipMemcpy(buffer1, idata, n * sizeof(int), hipMemcpyHostToDevice);

            dim3 fullBlockPerGrid = ((n + blockSize - 1) / blockSize);
            int depth = ilog2ceil(n);

            timer().startGpuTimer();

            for (int d = 1; d <= depth; d++) {
                scan_one_iteration<<< fullBlockPerGrid, blockSize>>> (n, d, buffer2, buffer1);
                std::swap(buffer1, buffer2);
            }

            odata[0] = 0;
            hipMemcpy(odata + 1, buffer1, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);

            timer().endGpuTimer();

            hipFree(buffer1);
            hipFree(buffer2);
        }
    }
}
