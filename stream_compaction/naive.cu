#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include <hip/hip_runtime.h>
#include <>

constexpr int blockSize = 512;

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__

        __global__ void kernPartialSum(int N, int d, int* odata, const int *idata)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= N) { return; }
            if (index >= (1 << d))
            {
                odata[index] = idata[index - (1 << d)] + idata[index];
            }
            else
            {
                odata[index] = idata[index];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO

            // Ping-pong device data buffers
            int* dev_idata;
            int* dev_odata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(dev_odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToDevice);

            dim3 threadsPerBlock(blockSize);
            dim3 numBlocks((n + blockSize - 1) / blockSize);

            for (int d = 0; d < ilog2ceil(n); ++d)
            {
                kernPartialSum <<<numBlocks, threadsPerBlock>>> (n, d, dev_odata, dev_idata);
                checkCUDAError("Kernel launch failed.");
                std::swap(dev_odata, dev_idata);
            }

            odata[0] = 0;
            hipMemcpy(odata + 1, dev_idata, (n - 1) * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_odata);
            timer().endGpuTimer();
        }
    }
}
