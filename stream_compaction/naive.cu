#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"
#include ""

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        // TODO: __global__
        __global__ void kernHandleNonPower(int n, int d, int* buffer) {
            int index = threadIdx.x + blockIdx.x * blockDim.x;
            int pow2tod = 1 << d;

            if (index >= n - pow2tod) return;

            buffer[pow2tod + index] += buffer[index];
        }

        __global__ void kernNaiveScanStep(int n, int d, const int* readBuffer, int* writeBuffer) {
            // compute thread index
            int index = threadIdx.x + blockIdx.x * blockDim.x;
            if (index >= n) return;

            int pow2tod = 1 << d;

            if (index > pow2tod) {
                writeBuffer[index] = readBuffer[index] + readBuffer[index - pow2tod];
            }
            else {
                writeBuffer[index] = readBuffer[index];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int blockSize = 128;
            dim3 fullBlocksPerGrid{ (unsigned int) (n + blockSize - 1) / blockSize };

            int* dev_buffer1;
            int* dev_buffer2;

            hipMalloc((void**)&dev_buffer1, n * sizeof(int));
            checkCUDAError("hipMalloc dev_buffer1 failed!");
            hipMalloc((void**)&dev_buffer2, n * sizeof(int));
            checkCUDAError("hipMalloc dev_buffer2 failed!");

            hipMemcpy(dev_buffer2, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            StreamCompaction::Common::shiftArrayElements<<<fullBlocksPerGrid, blockSize>>>(n, 1, dev_buffer2, dev_buffer1);
            checkCUDAError("shiftArrayElements failed!");
            hipDeviceSynchronize();

            for (int d = 0; d < ilog2(n); ++d) {
                kernNaiveScanStep <<<fullBlocksPerGrid, blockSize>>>(n, d, dev_buffer1, dev_buffer2);
                checkCUDAError("naiveScanStep failed!");
                hipDeviceSynchronize();

                std::swap(dev_buffer1, dev_buffer2);
            }
            // perform last step 
            if ((1 << ilog2(n)) != n) {
                fullBlocksPerGrid.x = (n - (1 << ilog2(n)) + blockSize - 1) / blockSize;
                kernHandleNonPower<<<fullBlocksPerGrid, blockSize>>>(n, ilog2(n), dev_buffer1);
                checkCUDAError("handleNonPower failed!");
                hipDeviceSynchronize();
            }
            timer().endGpuTimer();

            hipMemcpy(odata, dev_buffer1, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_buffer1);
            hipFree(dev_buffer2);
        }
    }
}