#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernNaiveScan(int n, int d, int* odata, const int* idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            int offset = 1 << (d - 1);
            if (index >= offset) {
                odata[index] = idata[index - offset] + idata[index];
            }
            else {
                odata[index] = idata[index];
            }
        }

        void ChangeToExclusive(int n, int* odata) {
            for (int i = n - 1; i > 0; i--) {
                odata[i] = odata[i - 1];
            }
            odata[0] = 0;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_odata;
            int blockSize = 128;
            int fullBlocksPerGrid = (n + blockSize - 1) / blockSize;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");

            // copy the input to GPU
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // outer loop
            for (int d = 1; d <= ilog2ceil(n); d++) {
                // parallel process
                kernNaiveScan<<<fullBlocksPerGrid, blockSize>>> (n, d, dev_odata, dev_idata);
                std::swap(dev_odata, dev_idata);
            }
            timer().endGpuTimer();

            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            // shift right and add 0 to the beginning to acquire the exclusive scan
            ChangeToExclusive(n, odata);

            // free memory
            hipFree(dev_idata);
            hipFree(dev_odata);
        }
    }
}
