#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "common.h"
#include "efficient_optimized.h"
#include <hip/device_functions.h>

// https://developer.nvidia.com/gpugems/gpugems3/part-vi-gpu-computing/chapter-39-parallel-prefix-sum-scan-cuda
// https://github.com/mark-poscablo/gpu-prefix-sum/tree/master
namespace StreamCompaction {
    namespace EfficientOptimized {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        void __global__ kernUpSweep(int n, int d, int* idata)
        {
            unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
            unsigned int offset = (unsigned int)(1 << (d + 1));
            if (index >= (unsigned int)(n >> (d + 1))) return;
            idata[(index + 1) * offset - 1] = idata[(index + 1) * offset - 1] + idata[(index + 1) * offset - 1 - (offset >> 1)];
        }

        void __global__ kernDownSweep(int n, int d, int* idata)
        {
            unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
            unsigned int offset = (unsigned int)(n >> d);
            if (index >= (unsigned int)(1 << d)) return;
            if (d == 0 && index == 0) idata[n - 1] = 0;
            int temp = idata[(index + 1) * offset - 1];
            idata[(index + 1) * offset - 1] += idata[(index + 1) * offset - 1 - (offset >> 1)];
            idata[(index + 1) * offset - 1 - (offset >> 1)] = temp;
        }

        __global__
        void gpu_add_block_sums(int* const d_out,
                const int* const d_in,
                int* const d_block_sums,
                const size_t numElems)
        {
            //unsigned int glbl_t_idx = blockDim.x * blockIdx.x + threadIdx.x;
            unsigned int d_block_sum_val = d_block_sums[blockIdx.x];

            //unsigned int d_in_val_0 = 0;
            //unsigned int d_in_val_1 = 0;

            // Simple implementation's performance is not significantly (if at all)
            //  better than previous verbose implementation
            unsigned int cpy_idx = 2 * blockIdx.x * blockDim.x + threadIdx.x;
            if (cpy_idx < numElems)
            {
                d_out[cpy_idx] = d_in[cpy_idx] + d_block_sum_val;
                if (cpy_idx + blockDim.x < numElems)
                    d_out[cpy_idx + blockDim.x] = d_in[cpy_idx + blockDim.x] + d_block_sum_val;
            }
        }

        // Modified version of Mark Harris' implementation of the Blelloch scan
        // according to https://www.mimuw.edu.pl/~ps209291/kgkp/slides/scan.pdf
        __global__
        void gpu_prescan(int* const d_out,
                int* const d_in,
                int* const d_block_sums,
                const unsigned int len,
                const unsigned int shmem_sz,
                const unsigned int max_elems_per_block)
        {
            // Allocated on invocation
            extern __shared__ unsigned int s_out[];

            int thid = threadIdx.x;
            int ai = thid;
            int bi = thid + blockDim.x;

            // Zero out the shared memory
            // Helpful especially when input size is not power of two
            s_out[thid] = 0;
            s_out[thid + blockDim.x] = 0;
            // If CONFLICT_FREE_OFFSET is used, shared memory
            //  must be a few more than 2 * blockDim.x
            if (thid + max_elems_per_block < shmem_sz)
                s_out[thid + max_elems_per_block] = 0;

            __syncthreads();

            // Copy d_in to shared memory
            // Note that d_in's elements are scattered into shared memory
            //  in light of avoiding bank conflicts
            unsigned int cpy_idx = max_elems_per_block * blockIdx.x + threadIdx.x;
            if (cpy_idx < len)
            {
                s_out[ai + CONFLICT_FREE_OFFSET(ai)] = d_in[cpy_idx];
                if (cpy_idx + blockDim.x < len)
                    s_out[bi + CONFLICT_FREE_OFFSET(bi)] = d_in[cpy_idx + blockDim.x];
            }

            // For both upsweep and downsweep:
            // Sequential indices with conflict free padding
            //  Amount of padding = target index / num banks
            //  This "shifts" the target indices by one every multiple
            //   of the num banks
            // offset controls the stride and starting index of 
            //  target elems at every iteration
            // d just controls which threads are active
            // Sweeps are pivoted on the last element of shared memory

            // Upsweep/Reduce step
            int offset = 1;
            for (int d = max_elems_per_block >> 1; d > 0; d >>= 1)
            {
                __syncthreads();

                if (thid < d)
                {
                    int ai = offset * ((thid << 1) + 1) - 1;
                    int bi = offset * ((thid << 1) + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);

                    s_out[bi] += s_out[ai];
                }
                offset <<= 1;
            }

            // Save the total sum on the global block sums array
            // Then clear the last element on the shared memory
            if (thid == 0)
            {
                d_block_sums[blockIdx.x] = s_out[max_elems_per_block - 1
                    + CONFLICT_FREE_OFFSET(max_elems_per_block - 1)];
                s_out[max_elems_per_block - 1
                    + CONFLICT_FREE_OFFSET(max_elems_per_block - 1)] = 0;
            }

            // Downsweep step
            for (int d = 1; d < max_elems_per_block; d <<= 1)
            {
                offset >>= 1;
                __syncthreads();

                if (thid < d)
                {
                    int ai = offset * ((thid << 1) + 1) - 1;
                    int bi = offset * ((thid << 1) + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);

                    unsigned int temp = s_out[ai];
                    s_out[ai] = s_out[bi];
                    s_out[bi] += temp;
                }
            }
            __syncthreads();

            // Copy contents of shared memory to global memory
            if (cpy_idx < len)
            {
                d_out[cpy_idx] = s_out[ai + CONFLICT_FREE_OFFSET(ai)];
                if (cpy_idx + blockDim.x < len)
                    d_out[cpy_idx + blockDim.x] = s_out[bi + CONFLICT_FREE_OFFSET(bi)];
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata, bool startTimer, bool isHost)
        {
            int* dev_odata;
            int* dev_idata;

            unsigned int blockSize = MAX_BLOCK_SIZE / 2;
            unsigned int maxElemsPerBlock = 2 * blockSize;

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMemset(dev_odata, 0, n * sizeof(unsigned int));
            checkCUDAError("hipMemset dev_odata failed!");

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMemset(dev_idata, 0, n * sizeof(unsigned int));
            checkCUDAError("hipMemset dev_idata failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), isHost ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy dev_idata failed!");

            unsigned int gridSize = (n + maxElemsPerBlock - 1) / maxElemsPerBlock;
            
            // Conflict free padding requires that shared memory be more than 2 * block_sz
            unsigned int shmemSize = maxElemsPerBlock + ((maxElemsPerBlock - 1) >> LOG_NUM_BANKS);

            // Allocate memory for array of total sums produced by each block
            // Array length must be the same as number of blocks
            int* d_block_sums;
            hipMalloc(&d_block_sums, sizeof(unsigned int) * gridSize);
            checkCUDAError("hipMalloc d_block_sums failed!");
            hipMemset(d_block_sums, 0, sizeof(unsigned int) * gridSize);
            checkCUDAError("hipMemset d_block_sums failed!");

            int gridSizeSums = (gridSize + maxElemsPerBlock - 1) / maxElemsPerBlock;

            int* d_dummy_blocks_sums;
            hipMalloc(&d_dummy_blocks_sums, sizeof(unsigned int) * gridSizeSums);
            checkCUDAError("hipMalloc d_dummy_blocks_sums failed!");
            hipMemset(d_dummy_blocks_sums, 0, sizeof(unsigned int) * gridSizeSums);
            checkCUDAError("hipMemset d_dummy_blocks_sums failed!");

            if (startTimer)
            {
                timer().startGpuTimer();
            }

            gpu_prescan << <gridSize, blockSize, sizeof(unsigned int)* shmemSize >> > (
                dev_odata,
                dev_idata,
                d_block_sums,
                n,
                shmemSize,
                maxElemsPerBlock);

            // compute prefix sum of sums array
            // sums:
            // [data[0] + .... + data[511], data[512] + ... + data[1023], ....]
            // ---->
            // [data[0] + .... + data[511], data[0] + ... + data[1023], ....]
            if (gridSize <= maxElemsPerBlock)
            {
                gpu_prescan << <1, blockSize, sizeof(unsigned int)* shmemSize >> > (
                    d_block_sums,
                    d_block_sums,
                    d_dummy_blocks_sums,
                    gridSize,
                    shmemSize,
                    maxElemsPerBlock);
            }
            else
            {
                int* d_in_block_sums;
                hipMalloc(&d_in_block_sums, sizeof(unsigned int) * gridSize);
                checkCUDAError("hipMalloc d_in_block_sums failed!");
                hipMemcpy(d_in_block_sums, d_block_sums, sizeof(unsigned int) * gridSize, hipMemcpyDeviceToDevice);
                checkCUDAError("hipMemcpy d_in_block_sums failed!");

                scan(gridSize, d_block_sums, d_in_block_sums, false, false);

                hipFree(d_in_block_sums);
                checkCUDAError("hipFree d_in_block_sums failed!");
            }

            gpu_add_block_sums << <gridSize, blockSize>> > (dev_odata, dev_odata, d_block_sums, n);

            if (startTimer)
            {
                timer().endGpuTimer();
            }

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_odata);
            checkCUDAError("hipFree dev_odata failed!");
            hipFree(dev_idata);
            checkCUDAError("hipFree dev_idata failed!");
            hipFree(d_block_sums);
            checkCUDAError("hipFree d_block_sums failed!");
            hipFree(d_dummy_blocks_sums);
            checkCUDAError("hipFree d_dummy_blocks_sums failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) 
        {
            int* dev_odata;
            int* dev_idata;
            int* dev_mapdata;
            int* dev_scan;

            unsigned int blockSize = MAX_BLOCK_SIZE / 2;
            unsigned int maxElemsPerBlock = 2 * blockSize;

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMemset(dev_odata, 0, n * sizeof(unsigned int));
            checkCUDAError("hipMemset dev_odata failed!");

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMemset(dev_idata, 0, n * sizeof(unsigned int));
            checkCUDAError("hipMemset dev_idata failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_idata failed!");

            hipMalloc((void**)&dev_mapdata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_mapdata failed!");

            hipMalloc((void**)&dev_scan, n * sizeof(int));
            checkCUDAError("hipMalloc dev_scan failed!");
            hipMemset(dev_scan, 0, n * sizeof(int));
            checkCUDAError("hipMemset idata to dev_odata failed!");

            unsigned int gridSize = (n + maxElemsPerBlock - 1) / maxElemsPerBlock;

            // Conflict free padding requires that shared memory be more than 2 * block_sz
            unsigned int shmemSize = maxElemsPerBlock + ((maxElemsPerBlock - 1) >> LOG_NUM_BANKS);

            // Allocate memory for array of total sums produced by each block
            // Array length must be the same as number of blocks
            int* d_block_sums;
            hipMalloc(&d_block_sums, sizeof(unsigned int) * gridSize);
            checkCUDAError("hipMalloc d_block_sums failed!");
            hipMemset(d_block_sums, 0, sizeof(unsigned int) * gridSize);
            checkCUDAError("hipMemset d_block_sums failed!");

            int gridSizeSums = (gridSize + maxElemsPerBlock - 1) / maxElemsPerBlock;

            int* d_dummy_blocks_sums;
            hipMalloc(&d_dummy_blocks_sums, sizeof(unsigned int) * gridSizeSums);
            checkCUDAError("hipMalloc d_dummy_blocks_sums failed!");
            hipMemset(d_dummy_blocks_sums, 0, sizeof(unsigned int) * gridSizeSums);
            checkCUDAError("hipMemset d_dummy_blocks_sums failed!");

            dim3 blocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

            timer().startGpuTimer();

            StreamCompaction::Common::kernMapToBoolean << < blocksPerGrid, BLOCK_SIZE >> > (n, dev_mapdata, dev_idata);
            hipMemcpy(dev_scan, dev_mapdata, n * sizeof(int), hipMemcpyDeviceToDevice);

#pragma region LocalMemOptimizedScan
            gpu_prescan << <gridSize, blockSize, sizeof(unsigned int)* shmemSize >> > (
                dev_scan,
                dev_scan,
                d_block_sums,
                n,
                shmemSize,
                maxElemsPerBlock);

            // compute prefix sum of sums array
            // sums:
            // [data[0] + .... + data[511], data[512] + ... + data[1023], ....]
            // ---->
            // [data[0] + .... + data[511], data[0] + ... + data[1023], ....]
            if (gridSize <= maxElemsPerBlock)
            {
                gpu_prescan << <1, blockSize, sizeof(unsigned int)* shmemSize >> > (
                    d_block_sums,
                    d_block_sums,
                    d_dummy_blocks_sums,
                    gridSize,
                    shmemSize,
                    maxElemsPerBlock);
            }
            else
            {
                int* d_in_block_sums;
                hipMalloc(&d_in_block_sums, sizeof(unsigned int) * gridSize);
                checkCUDAError("hipMalloc d_in_block_sums failed!");
                hipMemcpy(d_in_block_sums, d_block_sums, sizeof(unsigned int) * gridSize, hipMemcpyDeviceToDevice);
                checkCUDAError("hipMemcpy d_in_block_sums failed!");

                scan(gridSize, d_block_sums, d_in_block_sums, false, false);

                hipFree(d_in_block_sums);
                checkCUDAError("hipFree d_in_block_sums failed!");
            }

            gpu_add_block_sums << <gridSize, blockSize >> > (dev_scan, dev_scan, d_block_sums, n);
#pragma endregion

            StreamCompaction::Common::kernScatter << < blocksPerGrid, BLOCK_SIZE >> > (n, dev_odata, dev_idata, dev_mapdata, dev_scan);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            unsigned int countNon0;
            hipMemcpy(&countNon0, dev_scan + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            countNon0 += idata[n - 1] ? 1 : 0;

            hipFree(dev_odata);
            checkCUDAError("hipFree dev_odata failed!");
            hipFree(dev_idata);
            checkCUDAError("hipFree dev_idata failed!");
            hipFree(d_block_sums);
            checkCUDAError("hipFree d_block_sums failed!");
            hipFree(d_dummy_blocks_sums);
            checkCUDAError("hipFree d_dummy_blocks_sums failed!");
            hipFree(dev_mapdata);
            checkCUDAError("hipFree dev_mapdata failed!");
            hipFree(dev_scan);
            checkCUDAError("hipFree dev_scan failed!");

            return countNon0;
        }
    }
}
