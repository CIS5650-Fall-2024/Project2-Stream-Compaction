#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int* A, int offset) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) {
                return;
            }
            idx *= offset;
            A[idx + offset - 1] += A[idx + offset / 2 - 1];
        }

        __global__ void kernDownSweep(int n, int* A, int offset) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) {
                return;
            }
            idx *= offset;

            int temp = A[idx + offset / 2 - 1];
            A[idx + offset / 2 - 1] = A[idx + offset - 1];
            A[idx + offset - 1] += temp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata, bool timeFlag) {

            unsigned int blockSize = 128;

            int padding = 1 << ilog2ceil(n);

            int* A;
            size_t arraySize = n * sizeof(int);
            size_t paddedSize = padding * sizeof(int);
            hipMalloc((void**)&A, paddedSize);
            checkCUDAError("hipMalloc A failed!");

            hipMemcpy(A, idata, arraySize, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            checkCUDAError("hipMemcpy idata to A failed!");

            hipMemset(A + n, 0, (paddedSize - arraySize));
            hipDeviceSynchronize();
            checkCUDAError("hipMemcpy padding A failed!");

            int numThreads = padding;

            if (timeFlag)
                timer().startGpuTimer();
            for (int i = 0; i < ilog2ceil(n); i++) {
                int offset = 1 << (i + 1);
                numThreads /= 2;
                dim3 fullBlocksPerGrid = ((numThreads + blockSize - 1) / blockSize);
                kernUpSweep << <fullBlocksPerGrid, blockSize >> > (padding, A, offset);
                hipDeviceSynchronize();
                checkCUDAError("kernUpSweep failed!");
            }

            // assign 0 to the root of the tree for Down-Sweep
            hipMemset(A + n - 1, 0, sizeof(int));
            hipDeviceSynchronize();
            checkCUDAError("hipMemset to A failed!");

            for (int i = ilog2ceil(n) - 1; i >= 0; i--) {
                int offset = 1 << (i + 1);
                numThreads *= 2;
                dim3 fullBlocksPerGrid = ((numThreads + blockSize - 1) / blockSize);
                kernDownSweep << <fullBlocksPerGrid, blockSize >> > (padding, A, offset);
                checkCUDAError("kernDownSweep failed!");
            }
            if (timeFlag)
                timer().endGpuTimer();

            hipMemcpy(odata, A, arraySize, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            checkCUDAError("hipMemcpy A to odata failed!");

            hipFree(A);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {

            unsigned int blockSize = 128;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            // Create device arrays
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indices;
            int padLen = 1 << ilog2ceil(n);
            size_t arraySize = n * sizeof(int);
            size_t paddedSize = padLen * sizeof(int);

            hipMalloc((void**)&dev_idata, arraySize);
            hipMalloc((void**)&dev_bools, paddedSize);
            hipMalloc((void**)&dev_indices, paddedSize);
            hipMalloc((void**)&dev_odata, arraySize);
            hipDeviceSynchronize();
            checkCUDAError("hipMalloc failed!");

            hipMemcpy(dev_idata, idata, arraySize, hipMemcpyHostToDevice);
            hipDeviceSynchronize();
            checkCUDAError("hipMemcpy idata failed!");

            hipMemset(dev_bools + n, 0, (paddedSize - arraySize));
            hipDeviceSynchronize();
            checkCUDAError("hipMemset dev_bools failed!");


            timer().startGpuTimer();

            StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_bools, dev_idata);

            scan(n, dev_indices, dev_bools, 0);
            StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);
            hipDeviceSynchronize();

            timer().endGpuTimer();
            hipMemcpy(odata, dev_odata, arraySize, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata to data failed!");

            int compactLen = 0;
            hipMemcpy(&compactLen, dev_indices + padLen - 1, sizeof(int), hipMemcpyHostToDevice);

            hipFree(dev_bools);
            hipFree(dev_indices);
            hipFree(dev_idata);
            hipFree(dev_odata);

            return compactLen;
        }
    }
}
