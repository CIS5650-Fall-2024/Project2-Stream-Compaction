#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		// up-sweep kernel
        __global__ void kernUpSweep(int n, int* odata, const int* idata, int t) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }
			// exclusive scan
			odata[index] = (index > 0) ? idata[index - 1] : 0;
			__syncthreads();
			// upsweep
            for (int d = 0; d <= t; ++d) {
                int offset = 1 << (d + 1);
				int ai = index + offset - 1;
				int bi = index + (offset / 2) - 1;
                if (index < n && (index % offset) == 0) {
                    odata[ai] += odata[bi];
                }

                __syncthreads();
            }
        }

        // down-sweep kernel
		__global__ void kernDownSweep(int n, int* odata, const int* idata, int t) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= 1 << (t + 1)) {
				return;
			}
			// exclusive scan
			odata[index] = (index > 0) ? idata[index - 1] : 0;
			__syncthreads();
			// downsweep
			if (index == 0) {
				odata[n - 1] = 0;
			}
			for (int d = t; d >= 0; --d) {
				int offset = 1 << (d + 1);
				int ai = index + offset - 1;
				int bi = index + (offset / 2) - 1;
				if (index < n && (index % offset) == 0) {
					int temp = odata[bi];
					odata[bi] = odata[ai];
					odata[ai] += temp;
				}

				__syncthreads();
			}
		}

		// up sweep + down aweep
		__global__ void kernScan(int n, int* odata, const int* idata, int t) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			int paddedSize = 1 << (t + 1);
			if (index >= paddedSize) {
				return;
			}
			// exclusive scan
			//odata[index] = (index > 0) ? idata[index - 1] : 0;
			//odata[index] = idata[index];
			odata[index] = (index >= n) ? 0 : idata[index];
			__syncthreads();
			// upsweep
			for (int d = 0; d <= t; ++d) {
				int offset = 1 << (d + 1);
				int ai = index + offset - 1;
				int bi = index + (offset / 2) - 1;
				if (index < paddedSize && (index % offset) == 0) {
					odata[ai] += odata[bi];
				}

				__syncthreads();
			}
			// downsweep
			if (index == 0) {
				odata[paddedSize - 1] = 0;
			}
			
			for (int d = t; d >= 0; --d) {
				int offset = 1 << (d + 1);
				int ai = index + offset - 1;
				int bi = index + (offset / 2) - 1;
				if (index < paddedSize && (index % offset) == 0) {
					int temp = odata[bi];
					odata[bi] = odata[ai];
					odata[ai] += temp;
				}

				__syncthreads();
			}
		}


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
			// call kernel
			int* dev_idata;
			int* dev_odata;
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            int t = ilog2ceil(n) - 1;
            printf("log2_n - 1: %d\n", t);
            //kernUpSweep << <1, n >> > (n, dev_odata, dev_idata, t);
			//kernDownSweep << <1, n >> > (n, dev_odata, dev_idata, t);
			kernScan << <1, n >> > (n, dev_odata, dev_idata, t);
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_idata);
			hipFree(dev_odata);




            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
