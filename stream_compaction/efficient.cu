#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int d, int* data) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int offset = 1 << (d + 1); // 2^(d+1)
            int pos = index * offset;

            if (pos >= n) {
                return;
            }

            data[pos + offset - 1] += data[pos + (offset >> 1) - 1];
        }

        __global__ void kernDownSweep(int n, int d, int* data) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int offset = 1 << (d + 1); // 2^(d+1)
            int pos = index * offset;

            if (pos >= n) {
                return;
            }

            int t = data[pos + (offset >> 1) - 1];
            data[pos + (offset >> 1) - 1] = data[pos + offset - 1];
            data[pos + offset - 1] += t;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int enlargedSize = 1 << ilog2ceil(n); // enlarge the size to the nearest power of 2
            int* dev_idata;
            int blockSize = 64;

            hipMalloc((void**)&dev_idata, enlargedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");

            // copy the input to GPU (size n data)
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // up-sweep
            for (int d = 0; d <= ilog2ceil(enlargedSize) - 1; d++) {
                int fullBlocksPerGrid = (enlargedSize / (1 << (d + 1)) + blockSize - 1) / blockSize;
                kernUpSweep<<<fullBlocksPerGrid, blockSize>>> (enlargedSize, d, dev_idata);
            }

            // down-sweep
            hipMemset(dev_idata + enlargedSize - 1, 0, sizeof(int));
            for (int d = ilog2ceil(enlargedSize) - 1; d >= 0; d--) {
                int fullBlocksPerGrid = (enlargedSize / (1 << (d + 1)) + blockSize - 1) / blockSize;
                kernDownSweep<<<fullBlocksPerGrid, blockSize>>> (enlargedSize, d, dev_idata);
            }
            timer().endGpuTimer();

            // copy the result to odata (size n data)
            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            // free memory
            hipFree(dev_idata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int* dev_idata;
            int* dev_odata;
            int* dev_temp;
            int* dev_scan;
            int blockSize = 128;
            int fullBlocksPerGrid = (n + blockSize - 1) / blockSize;

            int enlargedSize = 1 << ilog2ceil(n); // enlarge the size to the nearest power of 2

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_temp, enlargedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_temp failed!");
            hipMalloc((void**)&dev_scan, enlargedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_scan failed!");

            // copy the input to GPU
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // map
            StreamCompaction::Common::kernMapToBoolean<<<fullBlocksPerGrid, blockSize>>> (n, dev_temp, dev_idata);

            // scan (implemented again)
            hipMemcpy(dev_scan, dev_temp, n * sizeof(int), hipMemcpyDeviceToDevice);
            // up-sweep
            for (int d = 0; d <= ilog2ceil(enlargedSize) - 1; d++) {
                int fullBlocksPerGridEnlarged = (enlargedSize / (1 << (d + 1)) + blockSize - 1) / blockSize;
                kernUpSweep<<<fullBlocksPerGridEnlarged, blockSize>>> (enlargedSize, d, dev_scan);
            }

            // down-sweep
            hipMemset(dev_scan + enlargedSize - 1, 0, sizeof(int));
            for (int d = ilog2ceil(enlargedSize) - 1; d >= 0; d--) {
                int fullBlocksPerGridEnlarged = (enlargedSize / (1 << (d + 1)) + blockSize - 1) / blockSize;
                kernDownSweep<<<fullBlocksPerGridEnlarged, blockSize>>> (enlargedSize, d, dev_scan);
            }

            // scatter
            StreamCompaction::Common::kernScatter<<<fullBlocksPerGrid, blockSize>>> (n, dev_odata, dev_idata, dev_temp, dev_scan);
            timer().endGpuTimer();

            // calculate count
            int lastScanValue = 0;
            int lastTempValue = 0;
            hipMemcpy(&lastScanValue, dev_scan + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&lastTempValue, dev_temp + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            int count = lastScanValue + lastTempValue;

            // copy the result to odata
            hipMemcpy(odata, dev_odata, count * sizeof(int), hipMemcpyDeviceToHost);

            // free memory
            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_temp);
            hipFree(dev_scan);

            return count;
        }
    }
}
