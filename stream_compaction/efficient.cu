#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // Finds the modulo of value with 2^power)
        __device__ int modPowBase2(int value, int power)
        {
            int shift = (sizeof(int) * 8 - power);
            return (value << shift) >> shift;
        }

        // Performs parallel reduction
        __global__ void kernUpSweep(int n, int d, int *data)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) return;

            if (modPowBase2(index, d + 1)) return;

            data[index + (1 << (d + 1)) - 1] += data[index + (1 << d) - 1];
        }

        // Build a scan from an up-swept input
        __global__ void kernDownSweep(int n, int d, int* data)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n - 1) return;

            if (modPowBase2(index, d + 1)) return;

            int t = data[index + (1 << d) - 1];
            data[index + (1 << d) - 1] = data[index + (1 << (d + 1)) - 1];
            data[index + (1 << (d + 1)) - 1] += t;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            // Number of levels needed for the scan
            int maxDepth = ilog2ceil(n);
            // Padded size of the work device array
            size_t dataSize = (1ull << maxDepth);
            // Reset value for down-sweeping
            int zero = 0;
            
            // Allocate device arrays
            int *dev_data;

            hipMalloc((void**)&dev_data, dataSize * sizeof(int));
            checkCUDAError("hipMalloc Efficient::scan::dev_data failed!");

            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);

            dim3 gridDim((dataSize + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            
            // Up-Sweep
            for (int d = 0; d < maxDepth; ++d)
            {
                kernUpSweep<<<gridDim, blockSize>>>(dataSize, d, dev_data);
            }

            // Replace the last element with 0
            hipMemcpy(dev_data + dataSize - 1, &zero, sizeof(int), hipMemcpyHostToDevice);

            // Down-Sweep
            for (int d = maxDepth - 1; d >= 0; --d)
            {
                kernDownSweep<<<gridDim, blockSize>>>(dataSize, d, dev_data);
            }

            timer().endGpuTimer();

            // Copy the output data
            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

            // Free device arrays
            hipFree(dev_data);
            checkCUDAError("hipFree Efficient::scan failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            // Number of levels needed for the scan
            int maxDepth = ilog2ceil(n);
            // Padded size of the work device array
            size_t dataSize = (1ull << maxDepth);
            // Reset value for down-sweeping
            int zero = 0;

            // Allocate device arrays
            int *dev_idata, *dev_bools, *dev_indices, *dev_odata;

            hipMalloc((void**)&dev_idata, dataSize * sizeof(int));
            checkCUDAError("hipMalloc Efficient::compact::dev_idata failed!");

            hipMalloc((void**)&dev_bools, dataSize * sizeof(int));
            checkCUDAError("hipMalloc Efficient::compact::dev_bools failed!");

            hipMalloc((void**)&dev_indices, dataSize * sizeof(int));
            checkCUDAError("hipMalloc Efficient::compact::dev_indices failed!");

            hipMalloc((void**)&dev_odata, dataSize * sizeof(int));
            checkCUDAError("hipMalloc Efficient::compact::dev_odata failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            dim3 gridDim((dataSize + blockSize - 1) / blockSize);

            timer().startGpuTimer();

            // Map to boolean
            Common::kernMapToBoolean<<<gridDim, blockSize>>>(dataSize, dev_bools, dev_idata);


            // Scan the indices
            hipMemcpy(dev_indices, dev_bools, dataSize * sizeof(int), hipMemcpyHostToHost);

            // Up-Sweep
            for (int d = 0; d < maxDepth; ++d)
            {
                kernUpSweep<<<gridDim, blockSize>>>(dataSize, d, dev_indices);
            }

            // Replace the last element with 0
            hipMemcpy(dev_indices + dataSize - 1, &zero, sizeof(int), hipMemcpyHostToDevice);

            // Down-Sweep
            for (int d = maxDepth - 1; d >= 0; --d)
            {
                kernDownSweep<<<gridDim, blockSize>>>(dataSize, d, dev_indices);
            }


            // Scatter
            Common::kernScatter<<<gridDim, blockSize>>>(dataSize, dev_odata, dev_idata, dev_bools, dev_indices);

            timer().endGpuTimer();

            // Retrieve the output size
            int* ptr_size = new int();
            int* ptr_doLast = new int();
            hipMemcpy(ptr_size, dev_indices + dataSize - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(ptr_doLast, dev_bools + dataSize - 1, sizeof(int), hipMemcpyDeviceToHost);

            int size = *ptr_size + (*ptr_doLast ? 1 : 0);

            delete(ptr_doLast);
            delete(ptr_size);

            // Copy the output data
            hipMemcpy(odata, dev_odata, size * sizeof(int), hipMemcpyDeviceToHost);

            // Free device arrays
            hipFree(dev_odata);
            hipFree(dev_indices);
            hipFree(dev_bools);
            hipFree(dev_idata);
            checkCUDAError("hipFree Efficient::compact failed!");

            return size;
        }
    }
}
