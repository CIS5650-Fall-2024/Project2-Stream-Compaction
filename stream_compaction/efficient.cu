#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
		#define blockSize 256
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void kernUpSweep(int n, int* odata, int d) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n || index % (1 << (d + 1)) != 0) return;
            
			odata[index + (1 << (d + 1)) - 1] += odata[index + (1 << d) - 1];
		}

		__global__ void kernDownSweep(int n, int* odata, int d) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n || index % (1 << (d + 1)) != 0) return;


			int t = odata[index + (1 << d) - 1];
			odata[index + (1 << d) - 1] = odata[index + (1 << (d + 1)) - 1];
			odata[index + (1 << (d + 1)) - 1] += t;
		}

		__global__ void computeTempArray(int n, int* odata, const int* idata) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) return;

			odata[index] = idata[index] == 0 ? 0 : 1;
		}

		__global__ void scatter(int n, int* odata, const int* idata, const int* bools, const int* scan) {	
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) return;

			if (bools[index] > 0) {
				odata[scan[index]] = idata[index];
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO
			//int blockSize = 128;
			int npower2 = 1 << ilog2ceil(n);
			int* dev_odata;

			hipMalloc((void**)&dev_odata, npower2 * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMemset(dev_odata, 0, npower2 * sizeof(int));
			hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy idata to dev_idata failed!");

			dim3 fullBlocksPerGrid((npower2 + blockSize - 1) / blockSize);
			timer().startGpuTimer();

			// up sweep
			for (int d = 0; d < ilog2ceil(n); d++) {
				kernUpSweep << <fullBlocksPerGrid, blockSize >> > (npower2, dev_odata, d);
				checkCUDAError("kernUpSweep failed!");
				hipDeviceSynchronize();
			}

			// down sweep
			hipMemset(dev_odata + npower2 - 1, 0, sizeof(int));
			for (int d = ilog2ceil(npower2) - 1; d >= 0; d--) {
				kernDownSweep << <fullBlocksPerGrid, blockSize >> > (npower2, dev_odata, d);
				checkCUDAError("kernDownSweep failed!");
				hipDeviceSynchronize();
			}
			timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_odata to odata failed!");

			hipFree(dev_odata);

			/*for (int i = 0; i < n; i++) {
				printf("%d ", odata[i]);
			}*/
        }
        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        

		int compactPower2(int n, int* odata, const int* idata) {
			// TODO
			//int blockSize = 128;

			int* dev_tempArray;
			int* dev_scanArray;
			int* dev_idata;
			int* dev_odata;

			hipMalloc((void**)&dev_tempArray, n * sizeof(int));
			checkCUDAError("hipMalloc dev_tempArray failed!");
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			hipMalloc((void**)&dev_scanArray, n * sizeof(int));
			checkCUDAError("hipMalloc dev_scanArray failed!");

			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy idata to dev_idata failed!");
			timer().startGpuTimer();

			// compute tempArray
			computeTempArray << <(n + blockSize - 1) / blockSize, blockSize >> > (n, dev_tempArray, dev_idata);
			checkCUDAError("computeTempArray failed!");
			hipDeviceSynchronize();

			// up sweep and down sweep
			hipMemcpy(dev_scanArray, dev_tempArray, n * sizeof(int), hipMemcpyDeviceToDevice);
			for (int d = 0; d < ilog2ceil(n); d++) {
				kernUpSweep << <(n + blockSize - 1) / blockSize, blockSize >> > (n, dev_scanArray, d);
				checkCUDAError("kernUpSweep failed!");
				hipDeviceSynchronize();
			}

			
			hipMemset(dev_scanArray + n - 1, 0, sizeof(int));
			for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
				kernDownSweep << <(n + blockSize - 1) / blockSize, blockSize >> > (n, dev_scanArray, d);
				checkCUDAError("kernDownSweep failed!");
				hipDeviceSynchronize();
			}

			// scatter
			scatter << <(n + blockSize - 1) / blockSize, blockSize >> > (n, dev_odata, dev_idata, dev_tempArray, dev_scanArray);
			checkCUDAError("scatter failed!");
			hipDeviceSynchronize();
			timer().endGpuTimer();

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

			int* host_scanArray = new int[n];
			hipMemcpy(host_scanArray, dev_scanArray, n * sizeof(int), hipMemcpyDeviceToHost);
			int count = host_scanArray[n - 1];

			delete[] host_scanArray;
			hipFree(dev_tempArray);
			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(dev_scanArray);

			return count;
		}

		int compact(int n, int* odata, const int* idata) {
			int npower2 = 1 << ilog2ceil(n);
			int* idata_power2 = new int[npower2];
			memset(idata_power2, 0, npower2 * sizeof(int));
			memcpy(idata_power2, idata, n * sizeof(int));

			int* odata_power2 = new int[npower2];
			memset(odata_power2, 0, npower2 * sizeof(int));

			int count = compactPower2(npower2, odata_power2, idata_power2);
			memcpy(odata, odata_power2, count * sizeof(int));

			delete[] idata_power2;
			delete[] odata_power2;
			
			return count;
		}
    }
}
