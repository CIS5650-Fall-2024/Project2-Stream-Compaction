#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int* A, int offset) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;

            // Dividing the input into groups each of offset size
            if (idx >= n / offset) {
                return;
            }
            idx *= offset;
            A[idx + offset - 1] += A[idx + offset / 2 - 1];
        }

        __global__ void kernDownSweep(int n, int* A, int offset) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n/offset) {
                return;
            }
            idx *= offset;

            int temp = A[idx + offset / 2 - 1];
            A[idx + offset / 2 - 1] = A[idx + offset - 1];
            A[idx + offset - 1] += temp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata, bool timeFlag) {

            unsigned int blockSize = 128;

            int padding = 1 << ilog2ceil(n);

            int* dev_odata;
            size_t arraySize = n * sizeof(int);
            size_t paddedSize = padding * sizeof(int);
            hipMalloc((void**)&dev_odata, paddedSize);
            checkCUDAError("hipMalloc dev_odata failed!");

            hipMemcpy(dev_odata, idata, arraySize, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to dev_odata failed!");

            hipMemset(dev_odata + n, 0, (paddedSize - arraySize));
            checkCUDAError("hipMemcpy padding dev_odata failed!");

            int numThreads = padding;

            if (timeFlag)
                timer().startGpuTimer();
            for (int i = 0; i < ilog2ceil(n); i++) {
                int offset = 1 << (i + 1);
                numThreads /= 2;
                
                dim3 fullBlocksPerGrid = ((numThreads + blockSize - 1) / blockSize);
                kernUpSweep << <fullBlocksPerGrid, blockSize >> > (padding, dev_odata, offset);
                hipDeviceSynchronize();
                checkCUDAError("kernUpSweep failed!");
            }

            // assign 0 to the root of the tree for Down-Sweep
            hipMemset(dev_odata + padding - 1, 0, sizeof(int));
            hipDeviceSynchronize();
            checkCUDAError("hipMemset to dev_odata failed!");

            for (int i = ilog2ceil(n) - 1; i >= 0; i--) {
                int offset = 1 << (i + 1);
                numThreads *= 2;
                dim3 fullBlocksPerGrid = ((numThreads + blockSize - 1) / blockSize);
                kernDownSweep << <fullBlocksPerGrid, blockSize >> > (padding, dev_odata, offset);
                checkCUDAError("kernDownSweep failed!");
            }
            if (timeFlag)
                timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, arraySize, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            checkCUDAError("hipMemcpy dev_odata to odata failed!");

            hipFree(dev_odata);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {

            // Create device arrays
            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indices;
            int padding = 1 << ilog2ceil(n);
            size_t arraySize = n * sizeof(int);
            size_t paddedSize = padding * sizeof(int);

            hipMalloc((void**)&dev_idata, arraySize);
            hipDeviceSynchronize();
            checkCUDAError("cudaMalloc1 failed!");

            hipMalloc((void**)&dev_bools, paddedSize);
            hipDeviceSynchronize();
            checkCUDAError("cudaMalloc2 failed!");

            hipMalloc((void**)&dev_indices, paddedSize);
            hipMalloc((void**)&dev_odata, arraySize);
            hipDeviceSynchronize();
            checkCUDAError("hipMalloc failed!");

            hipMemcpy(dev_idata, idata, arraySize, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata failed!");

            hipMemset(dev_bools + n, 0, (paddedSize - arraySize));
            checkCUDAError("hipMemset dev_bools failed!");

            unsigned int blockSize = 128;
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            timer().startGpuTimer();

            StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_bools, dev_idata);

            scan(n, dev_indices, dev_bools, 0);
            StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);
            hipDeviceSynchronize();
            checkCUDAError("kernel calls failed!");

            timer().endGpuTimer();
            hipMemcpy(odata, dev_odata, arraySize, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_odata to odata failed!");

            // check if last element of idata is valid, by checking dev_bools. 
            // If yes, then its index is compactLen - 1. If not, its index is compactLen.
            int isLastElemValid;
            hipMemcpy(&isLastElemValid, dev_bools + n - 1, sizeof(int), hipMemcpyDeviceToHost);

            int lastElemIdx;
            hipMemcpy(&lastElemIdx, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);

            int compactLen = (isLastElemValid) ? lastElemIdx + 1 : lastElemIdx;

            hipFree(dev_bools);
            hipFree(dev_indices);
            hipFree(dev_idata);
            hipFree(dev_odata);

            return (compactLen) ? compactLen : -1;
        }
    }
}
