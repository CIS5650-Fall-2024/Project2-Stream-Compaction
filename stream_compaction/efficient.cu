#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

		__global__ void reduction(int n, int* idata, int d) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n || index % (1 << (d + 1)) != 0) return;

			idata[index + (1 << (d + 1)) - 1] += idata[index + (1 << d) - 1];
		}

		__global__ void kernUpSweep(int n, int* odata, int d) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n || index % (1 << (d + 1)) != 0) return;
            
			odata[index + (1 << (d + 1)) - 1] += odata[index + (1 << d) - 1];
		}

		__global__ void kernDownSweep(int n, int* odata, int d) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n || index % (1 << (d + 1)) != 0) return;


			int t = odata[index + (1 << d) - 1];
			odata[index + (1 << d) - 1] = odata[index + (1 << (d + 1)) - 1];
			odata[index + (1 << (d + 1)) - 1] += t;
		}

		__global__ void computeTempArray(int n, int* odata, const int* idata) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) return;

			odata[index] = idata[index] == 0 ? 0 : 1;
		}

		__global__ void scatter(int n, int* odata, const int* idata, const int* bools, const int* scan) {	
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index >= n) return;

			if (idata[index] != 0) {
				odata[scan[index]] = idata[index];
			}
		}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            //timer().startGpuTimer();
            // TODO
			int blockSize = 128;

			int* dev_odata;

			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");
			
			hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy idata to dev_idata failed!");

			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

			for (int d = 0; d < ilog2ceil(n); d++) {
				kernUpSweep << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, d);
				checkCUDAError("kernUpSweep failed!");
				hipDeviceSynchronize();
			}

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_odata to odata failed!");


			hipMemset(dev_odata + n - 1, 0, sizeof(int));
			for (int d = ilog2ceil(n) - 1; d >= 0; d--) {
				kernDownSweep << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, d);
				checkCUDAError("kernDownSweep failed!");
				hipDeviceSynchronize();
			}

			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_odata to odata failed!");

			hipFree(dev_odata);

			/*for (int i = 0; i < n; i++) {
				printf("%d ", odata[i]);
			}*/
            //timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        

		int compactPower2(int n, int* odata, const int* idata) {
			timer().startGpuTimer();
			// TODO
			int blockSize = 128;

			int* dev_tempArray;
			int* dev_idata;
			int* dev_odata;

			int* host_tempArray = new int[n];
			int* host_scanArray = new int[n];
			memset(host_tempArray, 0, n * sizeof(int));
			memset(host_scanArray, 0, n * sizeof(int));

			hipMalloc((void**)&dev_tempArray, n * sizeof(int));
			checkCUDAError("hipMalloc dev_tempArray failed!");
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_idata failed!");
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			checkCUDAError("hipMalloc dev_odata failed!");

			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy idata to dev_idata failed!");

			// compute tempArray
			computeTempArray << <(n + blockSize - 1) / blockSize, blockSize >> > (n, dev_tempArray, dev_idata);
			checkCUDAError("computeTempArray failed!");
			hipDeviceSynchronize();

			hipMemcpy(host_tempArray, dev_tempArray, n * sizeof(int), hipMemcpyDeviceToHost);
			checkCUDAError("hipMemcpy dev_tempArray to host_tempArray failed!");

			// up sweep and down sweep
			scan(n, host_scanArray, host_tempArray);

			// scatter
			int* dev_scanArray;
			hipMalloc((void**)&dev_scanArray, n * sizeof(int));
			checkCUDAError("hipMalloc dev_scanArray failed!");
			hipMemcpy(dev_scanArray, host_scanArray, n * sizeof(int), hipMemcpyHostToDevice);
			checkCUDAError("hipMemcpy host_scanArray to dev_scanArray failed!");

			scatter << <(n + blockSize - 1) / blockSize, blockSize >> > (n, dev_odata, dev_idata, dev_tempArray, dev_scanArray);
			checkCUDAError("scatter failed!");
			hipDeviceSynchronize();
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);

			timer().endGpuTimer();

			hipFree(dev_tempArray);
			hipFree(dev_idata);


			return host_scanArray[n - 1];
		}

		int compact(int n, int* odata, const int* idata) {
			int npower2 = 1 << ilog2ceil(n);
			int* idata_power2 = new int[npower2];
			memset(idata_power2, 0, npower2 * sizeof(int));
			memcpy(idata_power2, idata, n * sizeof(int));

			int* odata_power2 = new int[npower2];
			memset(odata_power2, 0, npower2 * sizeof(int));

			int count = compactPower2(npower2, odata_power2, idata_power2);
			memcpy(odata, odata_power2, count * sizeof(int));

			delete[] idata_power2;
			delete[] odata_power2;
			
			return count;
		}
    }
}
