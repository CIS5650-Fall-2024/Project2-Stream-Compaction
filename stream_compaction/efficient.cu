#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 32
#define RECURSIVE_SCAN 0

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


		// up-sweep kernel
        __global__ void kernUpSweep(int n, int* odata, int* idata, int t) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }
#if RECURSIVE_SCAN
			// exclusive scan
			odata[index] = (index > 0) ? idata[index - 1] : 0;
			__syncthreads();
			// upsweep
            for (int d = 0; d <= t; ++d) {
                int offset = 1 << (d + 1);
				int ai = index + offset - 1;
				int bi = index + (offset / 2) - 1;
                if (index < n && (index % offset) == 0) {
                    odata[ai] += odata[bi];
                }

                __syncthreads();
            }
#else
			int offset = 1 << (t + 1); // 2^(d + 1)
			int ai = index + offset - 1;
			int bi = index + (offset / 2) - 1;
			if (index < n && ((index) % offset) == 0) {
				idata[ai] += idata[bi];
			}
#endif
		}

        // down-sweep kernel
		__global__ void kernDownSweep(int n, int* odata, const int* idata, int t) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
#if RECURSIVE_SCAN
			if (index >= 1 << (t + 1)) {
				return;
			}
			// exclusive scan
			odata[index] = (index > 0) ? idata[index - 1] : 0;
			__syncthreads();
			// downsweep
			if (index == 0) {
				odata[n - 1] = 0;
			}
			for (int d = t; d >= 0; --d) {
				int offset = 1 << (d + 1);
				int ai = index + offset - 1;
				int bi = index + (offset / 2) - 1;
				if (index < n && (index % offset) == 0) {
					int temp = odata[bi];
					odata[bi] = odata[ai];
					odata[ai] += temp;
				}

				__syncthreads();
			}
#else	
			if (index >= n) {
				return;
			}

			__syncthreads();			

			int offset = 1 << (t + 1);
			int ai = index + offset - 1;
			int bi = index + (offset / 2) - 1;
			if (index % offset == 0) {
				int temp = odata[bi];
				odata[bi] = odata[ai];
				odata[ai] += temp;
			}

			__syncthreads();


#endif 

		}

		// up sweep + down aweep
		__global__ void kernScan(int n, int* odata, const int* idata, int t) {
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			int paddedSize = 1 << (t + 1);
			if (index >= paddedSize) {
				return;
			}
			// exclusive scan
			//odata[index] = (index > 0) ? idata[index - 1] : 0;
			//odata[index] = idata[index];
			odata[index] = (index >= n) ? 0 : idata[index];
			__syncthreads();
			// upsweep
			for (int d = 0; d <= t; ++d) {
				int offset = 1 << (d + 1);
				int ai = index + offset - 1;
				int bi = index + (offset / 2) - 1;
				if (index < paddedSize && (index % offset) == 0) {
					odata[ai] += odata[bi];
				}

				__syncthreads();
			}
			// downsweep
			if (index == 0) {
				odata[paddedSize - 1] = 0;
			}
			
			for (int d = t; d >= 0; --d) {
				int offset = 1 << (d + 1);
				int ai = index + offset - 1;
				int bi = index + (offset / 2) - 1;
				if (index < paddedSize && (index % offset) == 0) {
					int temp = odata[bi];
					odata[bi] = odata[ai];
					odata[ai] += temp;
				}

				__syncthreads();
			}
		}


        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            // TODO
			int t = ilog2ceil(n) - 1;
			int peddedSize = 1 << (t + 1);
			//const int blockSize = 128;
			int numBlocks = (peddedSize + blockSize - 1) / blockSize;
			dim3 fullBlocksPerGrid(numBlocks);
			
			printf("log2_n - 1: %d\n", t);
			printf("array size: %d; pedded size: %d\n", n, peddedSize);
			printf("block numbers: %d\n", numBlocks);
			// call kernel
			int* dev_idata;
			int* dev_odata;
			hipMalloc((void**)&dev_idata, peddedSize * sizeof(int));
			hipMalloc((void**)&dev_odata, peddedSize * sizeof(int));
			hipMemset(dev_odata, 0, peddedSize * sizeof(int));
			hipMemset(dev_idata, 0, peddedSize * sizeof(int));
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(dev_odata, idata, n * sizeof(int), hipMemcpyHostToDevice);
           
			timer().startGpuTimer();

#if RECURSIVE_SCAN
            //kernUpSweep << <1, n >> > (n, dev_odata, dev_idata, t);
			//kernDownSweep << <1, n >> > (n, dev_odata, dev_idata, t);
			//kernScan << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, t); // arbitrary block size
			kernScan << <1, n >> > (n, dev_odata, dev_idata, t);
#else
			
			// up-sweep
			for (int d = 0; d <= t; ++d) {
				int offset = 1 << (d + 1);
				//int numBlocks = (n + offset - 1) / offset;
				//dim3 fullBlocksPerGrid(numBlocks);
				kernUpSweep << <numBlocks, blockSize >> > (peddedSize, dev_odata, dev_idata, d);
			}
			// down sweep
			// set last element to 0
			hipMemset(dev_idata + peddedSize - 1, 0, sizeof(int));
			for (int d = t; d >= 0; d--) {
				int offset = 1 << (d + 1);
				//int numBlocks = (n + offset - 1) / offset;
				//dim3 fullBlocksPerGrid(numBlocks);				
				kernDownSweep << <numBlocks, blockSize >> > (peddedSize, dev_idata, dev_idata, d);
			}

#endif
			timer().endGpuTimer();
			hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_idata);
			hipFree(dev_odata);

            
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            //timer().startGpuTimer();
            // TODO
			// compute bool array
			int t = ilog2ceil(n) - 1;
			int peddedSize = 1 << (t + 1);
			int* dev_bools;
			int* dev_idata;
			int* dev_indices;
			int* dev_odata;
			int* bools = new int[n];
			int* indices = new int[n];
			hipMalloc((void**)&dev_bools, n * sizeof(int));
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			hipMalloc((void**)&dev_indices, n * sizeof(int));
			hipMalloc((void**)&dev_odata, n * sizeof(int));
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
			hipMemset(dev_bools, 0, n * sizeof(int));
			StreamCompaction::Common::kernMapToBoolean << <1, n >> > (n, dev_bools, dev_idata);
			// scan
			//kernScan << <1, n >> > (n, dev_indices, dev_bools, t);
			// up-sweep
			int* temp = new int[n];
			temp = dev_bools;
			for (int i = 0; i <= t; i++) {
				int offset = 1 << (i + 1);
				int numBlocks = (n + offset - 1) / offset;
				dim3 fullBlocksPerGrid(numBlocks);
				kernUpSweep << <fullBlocksPerGrid, blockSize >> > (n, temp, temp, i);
			}

			// set last element to 0
			hipMemset(temp + n - 1, 0, sizeof(int));
			// down-sweep
			for (int i = t; i >= 0; i--) {
				int offset = 1 << (i + 1);
				int numBlocks = (n + offset - 1) / offset;
				dim3 fullBlocksPerGrid(numBlocks);
				kernDownSweep << <fullBlocksPerGrid, blockSize >> > (n, temp, temp, i);
			}
			dev_indices = temp;
			// scatter
			StreamCompaction::Common::kernScatter << <1, n >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);
			hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(bools, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(indices, dev_indices, n * sizeof(int), hipMemcpyDeviceToHost);
			int count = bools[n - 1] ? indices[n - 1] + 1 : indices[n - 1];
			hipFree(dev_bools);
			hipFree(dev_idata);
			hipFree(dev_indices);
			hipFree(dev_odata);
			delete[] bools;
			delete[] indices;
			delete[] temp;


            //timer().endGpuTimer();
            return count;
        }
    }
}
