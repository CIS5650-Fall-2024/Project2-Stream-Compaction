#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""

#define BLOCKSIZE 128

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void kernUpSweep(int n, int i, int* data) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx < n) {
                int k1 = 1 << i + 1;
                int k2 = 1 << i;
                if ((idx & (k1 - 1)) == 0) {
                    data[idx + k1 - 1] += data[idx + k2 - 1];
                }
            }
        }
        __global__ void kernDownSweep(int n, int i, int* data) {
            int idx = threadIdx.x + (blockIdx.x * blockDim.x);
            if (idx < n) {
                int k1 = 1 << i + 1;
                int k2 = 1 << i;
                if ((idx & (k1 - 1)) == 0) {
                    int t = data[idx + k2 - 1];
                    data[idx + k2 - 1] = data[idx + k1 - 1];
                    data[idx + k1 - 1] += t;
                }
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int len = 1 << ilog2ceil(n);
            dim3 blockPerGrid((len + BLOCKSIZE - 1) / BLOCKSIZE);
            int* dev_data;
            hipMalloc((void**)&dev_data, len * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_data failed!");
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("hipMemcpy dev_data failed!");

            timer().startGpuTimer();
            // up-sweep
            for (int i = 0; i <= ilog2ceil(n) - 1; ++i) {
                kernUpSweep << <blockPerGrid, BLOCKSIZE >> > (len, i, dev_data);
            }
            // set last element 0
            hipMemset(dev_data + len - 1, 0, sizeof(int));
            // down-sweep
            for (int i = ilog2ceil(n) - 1; i >= 0; --i) {
                kernDownSweep << <blockPerGrid, BLOCKSIZE >> > (len, i, dev_data);
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("hipMemcpy dev_dataToodata failed!");
            hipFree(dev_data);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int len = 1 << ilog2ceil(n);
            dim3 blockPerGrid((len + BLOCKSIZE - 1) / BLOCKSIZE);
            int* dev_odata;
            int* dev_idata;
            int* dev_bool;
            int* dev_indices;
            hipMalloc((void**)&dev_odata, len * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_idata, len * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_idata failed!");
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipMalloc((void**)&dev_bool, len * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_bool failed!");
            hipMalloc((void**)&dev_indices, len * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_indices failed!");

            
            timer().startGpuTimer();
            StreamCompaction::Common::kernMapToBoolean <<< blockPerGrid, BLOCKSIZE >> > (len, dev_bool, dev_idata);
            checkCUDAErrorFn("kernMapToBoolean failed!");
            hipMemcpy(dev_indices, dev_bool, len * sizeof(int), hipMemcpyDeviceToDevice);
            // up-sweep
            for (int i = 0; i <= ilog2ceil(n) - 1; ++i) {
                kernUpSweep << <blockPerGrid, BLOCKSIZE >> > (len, i, dev_indices);
            }
            // set last element 0
            hipMemset(dev_indices + len - 1, 0, sizeof(int));
            // down-sweep
            for (int i = ilog2ceil(n) - 1; i >= 0; --i) {
                kernDownSweep << <blockPerGrid, BLOCKSIZE >> > (len, i, dev_indices);
            }
            // scatter
            StreamCompaction::Common::kernScatter << <blockPerGrid, BLOCKSIZE >> > (len, dev_odata, dev_idata, dev_bool, dev_indices);
            checkCUDAErrorFn("kernScatter failed!");
            timer().endGpuTimer();
            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            int num = 0;
            hipMemcpy(&num, dev_indices + len - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_odata);
            hipFree(dev_idata);
            hipFree(dev_bool);
            hipFree(dev_indices);
            return num;
        }
    }
}
