#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int* A, int offset) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) {
                return;
            }
            idx *= offset;
            A[idx + offset - 1] += A[idx + offset / 2 - 1];
        }

        __global__ void kernDownSweep(int n, int* A, int offset) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) {
                return;
            }
            idx *= offset;

            int temp = A[idx + offset / 2 - 1];
            A[idx + offset / 2 - 1] = A[idx + offset - 1];
            A[idx + offset - 1] += temp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {

            unsigned int blockSize = 128;

            int padding = 1 << ilog2ceil(n);

            int* A;
            size_t arraySize = n * sizeof(int);
            size_t paddedSize = padding * sizeof(int);
            hipMalloc((void**)&A, paddedSize);
            checkCUDAError("hipMalloc A failed!");

            hipMemcpy(A, idata, arraySize, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to A failed!");

            hipMemcpy(A + n, 0, (paddedSize - arraySize), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to A failed!");
            hipDeviceSynchronize();

            timer().startGpuTimer();
            int numThreads = n;

            for (int i = 0; i < ilog2ceil(n); i++) {
                int offset = 1 << (i + 1);
                numThreads /= 2;
                dim3 fullBlocksPerGrid = ((numThreads + blockSize - 1) / blockSize);
                kernUpSweep << <fullBlocksPerGrid, blockSize >> > (n, A, offset);
                hipDeviceSynchronize();
            }

            // assign 0 to the root of the tree for Down-Sweep
            hipMemset(A + n - 1, 0, sizeof(int));

            for (int i = ilog2ceil(n) - 1; i >= 0; i--) {
                int offset = 1 << (i + 1);
                numThreads *= 2;
                dim3 fullBlocksPerGrid = ((numThreads + blockSize - 1) / blockSize);
                kernDownSweep << <fullBlocksPerGrid, blockSize >> > (n, A, offset);
            }
            timer().endGpuTimer();


            hipMemcpy(odata, A, arraySize, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy odata failed!");

            hipFree(A);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            timer().endGpuTimer();
            return -1;
        }
    }
}
