#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "common.h"
#include "efficient.h"

__device__ inline int twoPow(int d) {
    return (1 << (d));
}

inline int twoPow_Host(int d) {
    return (1 << (d));
}

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void upSweep(int n, int d, int* x) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) return;
            if (idx % twoPow(d + 1) != 0) return;
            x[idx + twoPow(d + 1) - 1] += x[idx + (twoPow(d)) - 1];
        }

        __global__ void downSweep(int n, int d, int* x) {
            int idx = blockDim.x * blockIdx.x + threadIdx.x;
            if (idx >= n) return;
            if (idx % twoPow(d + 1) != 0) return;

            int tmp = x[idx + twoPow(d) - 1];
            x[idx + twoPow(d) - 1] = x[idx + twoPow(d + 1) - 1];
            x[idx + twoPow(d + 1) - 1] += tmp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            // TODO
            int size = twoPow_Host(ilog2ceil(n));
            dim3 blockPerGrids((size + blockSize - 1) / blockSize);
            int* dev_idata;
            hipMalloc((void**)&dev_idata, size * sizeof(int));
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            hipDeviceSynchronize();

            timer().startGpuTimer();
            // UpSweep
            for (int d = 0; d < ilog2ceil(size); d++) {
                upSweep << <blockPerGrids, blockSize >> > (n, d, dev_idata);
                hipDeviceSynchronize();
            }
            hipMemset(dev_idata + size - 1, 0, sizeof(int));

            // DownSweep
            for (int d = ilog2ceil(size) - 1; d >= 0; d--) {
                downSweep << <blockPerGrids, blockSize >> > (n, d, dev_idata);
                hipDeviceSynchronize();
            }
            timer().endGpuTimer();

            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            
            int* dev_bools;
            int* dev_indices;
            int* dev_idata;
            int* dev_odata;
            int size = twoPow_Host(ilog2ceil(n));
            int cnt = 0;

            dim3 blockPerGrids((n + blockSize - 1) / blockSize);
            dim3 fullBlockPerGrids((size + blockSize - 1) / blockSize);

            hipMalloc((void**)&dev_bools, size * sizeof(int));
            hipMalloc((void**)&dev_indices, size * sizeof(int));
            hipMalloc((void**)&dev_idata, size * sizeof(int));
            hipMalloc((void**)&dev_odata, size * sizeof(int));
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            Common::kernMapToBoolean << <blockPerGrids, blockSize >> > (n, dev_bools, dev_idata);
            hipDeviceSynchronize();
            hipMemcpy(dev_indices, dev_bools, n * sizeof(int), hipMemcpyDeviceToDevice);

            // scan
            for (int d = 0; d < ilog2ceil(size); d++) {
                upSweep << <fullBlockPerGrids, blockSize >> > (n, d, dev_indices);
            }
            hipMemset(dev_indices + size - 1, 0, sizeof(int));

            for (int d = ilog2ceil(size) - 1; d >= 0; d--) {
                downSweep << <fullBlockPerGrids, blockSize >> > (n, d, dev_indices);
            }
            
            Common::kernScatter << <blockPerGrids, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);
            timer().endGpuTimer();
            hipMemcpy(&cnt, dev_indices + size - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(odata, dev_odata, cnt * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_indices);
            hipFree(dev_bools);
            

            return cnt;
        }
    }
}
