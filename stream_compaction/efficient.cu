#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernelEfficientScanUpSweep(int N, int d, int* data) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int d_2 = 1 << d;
            if (index < N && ((index + 1) % (d_2 * 2) == 0)) {
                data[index] += data[index - d_2];
            }
        }

        __global__ void kernelEfficientScanDownSweep(int N, int d, int* data) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            int d_2 = 1 << d;
            if (index < N && ((index + 1) % (d_2 * 2) == 0)) {
                int tmp = data[index];
                data[index] += data[index - d_2];
                data[index - d_2] = tmp;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // iteration initialization
            int iterNum = ilog2ceil(n);
            int tmpLength = 1 << iterNum;
            int blockNum((tmpLength + blockSize - 1) / blockSize);

            //device memory initialized
            int* dev_tmp;
            hipMalloc((void**)&dev_tmp, tmpLength * sizeof(int));
            checkCUDAError("hipMalloc dev_tmp failed!");  
            // copy array from cpu to gpu
            hipMemcpy(dev_tmp, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to dev_tmp failed!");
            // fill pad with 0
            if (tmpLength > n) {
                hipMemset(dev_tmp + n, 0, (tmpLength - n) * sizeof(int));
                checkCUDAError("hipMemset dev_tmp pad values failed!");
            }

            timer().startGpuTimer();
            
            // up sweep
            for (int d = 0; d < iterNum; d++) {
                kernelEfficientScanUpSweep<<<blockNum, blockSize>>>(tmpLength, d, dev_tmp);
                checkCUDAError("kernelEfficientScanUpSweep failed!");
            }
            hipDeviceSynchronize();

            //set root to 0
            hipMemset(dev_tmp + tmpLength - 1, 0, sizeof(int));
            checkCUDAError("hipMemset dev_tmp root to 0 failed!");

            // down sweep
            for (int d = iterNum - 1; d >= 0; d--) {
                kernelEfficientScanDownSweep<<<blockNum, blockSize>>>(tmpLength, d, dev_tmp);
                checkCUDAError("kernelEfficientScanDownSweep failed!");
            }

            timer().endGpuTimer();

            // copy array from gpu to cpu
            hipMemcpy(odata, dev_tmp, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy final odata failed!");

            // free memory
            hipFree(dev_tmp);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            // iteration initialization
            int iterNum = ilog2ceil(n);
            int tmpLength = 1 << iterNum;
            int originalBlockNum((n + blockSize - 1) / blockSize);
            int blockNum((tmpLength + blockSize - 1) / blockSize);

            //device memory initialized
            int* dev_idata;
            int* dev_bools;
            int* dev_indices;
            int* dev_odata;
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idate failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_bools, n * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed!");
            hipMalloc((void**)&dev_indices, tmpLength * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            // copy array from cpu to gpu
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata to dev_idate failed!");
            
            timer().startGpuTimer();
            
            // map
            Common::kernMapToBoolean<<<originalBlockNum, blockSize>>>(n, dev_bools, dev_idata);
            checkCUDAError("kernMapToBoolean failed!");

            //copy from dev_bools to dev_indices for scan
            hipMemcpy(dev_indices, dev_bools, n * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy dev_bools to dev_indices failed!");

            // fill pad with 0
            if (tmpLength > n) {
                hipMemset(dev_indices + n, 0, (tmpLength - n) * sizeof(int));
                checkCUDAError("hipMemset dev_indices pad values failed!");
            }

            // up sweep
            for (int d = 0; d < iterNum; d++) {
                kernelEfficientScanUpSweep<<<blockNum, blockSize>>>(tmpLength, d, dev_indices);
                checkCUDAError("kernelEfficientScanUpSweep failed!");
            }
            hipDeviceSynchronize();

            //set root to 0
            hipMemset(dev_indices + tmpLength - 1, 0, sizeof(int));
            checkCUDAError("hipMemset dev_indices root to 0 failed!");

            // down sweep
            for (int d = iterNum - 1; d >= 0; d--) {
                kernelEfficientScanDownSweep<<<blockNum, blockSize>>>(tmpLength, d, dev_indices);
                checkCUDAError("kernelEfficientScanDownSweep failed!");
            }

            //scatter
            Common::kernScatter<<<originalBlockNum, blockSize>>>(n, dev_odata, dev_idata, dev_bools, dev_indices);
            checkCUDAError("kernScatter failed!");

            timer().endGpuTimer();

            // get total non-zero count
            int count;
            hipMemcpy(&count, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            count += (int)(idata[n - 1] != 0);

            // copy array from gpu to cpu
            hipMemcpy(odata, dev_odata, count * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy final odata failed!");

            // free memory
            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_bools);
            hipFree(dev_indices);

            return count;
        }
    }
}
