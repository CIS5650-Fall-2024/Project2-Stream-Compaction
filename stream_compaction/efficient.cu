#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int d, int *data)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;

            if (index >= n) return;

            if (index % (1 << (d + 1))) return;

            data[index + (1 << (d + 1)) - 1] += data[index + (1 << d) - 1];
        }

        __global__ void kernDownSweep(int n, int d, int* data)
        {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;

            if (index >= n - 1) return;

            int test = (index % (1 << (d + 1)));
            if (index % (1 << (d + 1))) return;

            int t = data[index + (1 << d) - 1];
            data[index + (1 << d) - 1] = data[index + (1 << (d + 1)) - 1];
            data[index + (1 << (d + 1)) - 1] += t;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int *dev_data;

            int depth_max = ilog2ceil(n);
            size_t dataSize = (1ull << depth_max);

            hipMalloc((void**)&dev_data, dataSize * sizeof(int));
            checkCUDAError("hipMalloc Efficient::scan::dev_data failed!");

            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);

            dim3 gridDim((dataSize + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            
            for (int d = 0; d < depth_max; ++d)
            {
                kernUpSweep<<<gridDim, blockSize>>>(dataSize, d, dev_data);
            }

            int* zero = new int(0);
            hipMemcpy(dev_data + dataSize - 1, zero, sizeof(int), hipMemcpyHostToDevice);
            delete(zero);

            for (int d = depth_max - 1; d >= 0; --d)
            {
                kernDownSweep<<<gridDim, blockSize>>>(dataSize, d, dev_data);
            }

            timer().endGpuTimer();

            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_data);
            checkCUDAError("hipFree Efficient::scan failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int *dev_idata, *dev_bools, *dev_indices, *dev_odata;

            int depth_max = ilog2ceil(n);
            size_t dataSize = (1ull << depth_max);

            hipMalloc((void**)&dev_idata, dataSize * sizeof(int));
            checkCUDAError("hipMalloc Efficient::compact::dev_idata failed!");

            hipMalloc((void**)&dev_bools, dataSize * sizeof(int));
            checkCUDAError("hipMalloc Efficient::compact::dev_bools failed!");

            hipMalloc((void**)&dev_indices, dataSize * sizeof(int));
            checkCUDAError("hipMalloc Efficient::compact::dev_indices failed!");

            hipMalloc((void**)&dev_odata, dataSize * sizeof(int));
            checkCUDAError("hipMalloc Efficient::compact::dev_odata failed!");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            dim3 gridDim((dataSize + blockSize - 1) / blockSize);

            timer().startGpuTimer();

            // Map to boolean
            Common::kernMapToBoolean<<<gridDim, blockSize>>>(dataSize, dev_bools, dev_idata);
            hipMemcpy(dev_indices, dev_bools, dataSize * sizeof(int), hipMemcpyHostToHost);

            // Scan
            for (int d = 0; d < depth_max; ++d)
            {
                kernUpSweep<<<gridDim, blockSize>>>(dataSize, d, dev_indices);
            }

            int* zero = new int(0);
            hipMemcpy(dev_indices + dataSize - 1, zero, sizeof(int), hipMemcpyHostToDevice);
            delete(zero);

            for (int d = depth_max - 1; d >= 0; --d)
            {
                kernDownSweep<<<gridDim, blockSize>>>(dataSize, d, dev_indices);
            }

            // Scatter
            Common::kernScatter<<<gridDim, blockSize>>>(dataSize, dev_odata, dev_idata, dev_bools, dev_indices);

            timer().endGpuTimer();

            int* ptr_size = new int();
            int* ptr_doLast = new int();
            hipMemcpy(ptr_size, dev_indices + dataSize - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(ptr_doLast, dev_bools + dataSize - 1, sizeof(int), hipMemcpyDeviceToHost);

            int size = *ptr_size + (*ptr_doLast ? 1 : 0);

            delete(ptr_doLast);
            delete(ptr_size);

            hipMemcpy(odata, dev_odata, size * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_odata);
            hipFree(dev_indices);
            hipFree(dev_bools);
            hipFree(dev_idata);
            checkCUDAError("hipFree Efficient::compact failed!");

            return size;
        }
    }
}
