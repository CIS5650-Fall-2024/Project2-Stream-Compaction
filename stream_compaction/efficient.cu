#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <iostream>

#define globalIdx ((blockIdx.x * blockDim.x) + threadIdx.x)

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernEfficientScanMultiBlock(int n, int* data, int* sum) {
          extern __shared__ int temp[];
          temp[2 * threadIdx.x] = data[2 * globalIdx];
          temp[2 * threadIdx.x + 1] = data[2 * globalIdx + 1];

          // up sweep 
          for (int depth = 1; depth < n; depth <<= 1) {
            __syncthreads();
            int offset = threadIdx.x * (depth << 1);  // k * 2^(d+1)
            if (offset < n) {
              temp[offset + (depth << 1) - 1] += temp[offset + depth - 1];
            }
          }

          temp[n - 1] = 0;
          for (int depth = (n >> 1); depth >= 1; depth >>= 1) {
            __syncthreads();
            int offset = threadIdx.x * (depth << 1);
            if (offset < n) {
              int t = temp[offset + depth - 1];
              temp[offset + depth - 1] = temp[offset + (depth << 1) - 1];
              temp[offset + (depth << 1) - 1] += t;
            }
          }
          __syncthreads();

          // ensure we get an inclusive scan as our result
          if ((2 * threadIdx.x + 1) == n - 1) {
            data[2 * globalIdx] = temp[2 * threadIdx.x + 1];
            data[2 * globalIdx + 1] += temp[2 * threadIdx.x + 1];
            sum[blockIdx.x] = data[2 * globalIdx + 1];
            return; 
          }

          data[2 * globalIdx] = temp[2 * threadIdx.x + 1];
          data[2 * globalIdx + 1] = temp[2 * threadIdx.x + 2];
        }

        __global__ void kernEfficientScan(int n, int* data) {
          int idx = threadIdx.x;
          if (idx >= (n >> 1)) {
            return; 
          }
          extern __shared__ int temp[]; 
          temp[2 * idx]     = data[2 * idx];
          temp[2 * idx + 1] = data[2 * idx + 1];
          
          // up sweep 
          for (int depth = 1; depth < n; depth <<= 1) {
            __syncthreads();
            int offset = idx * (depth << 1);  // k * 2^(d+1)
            if (offset < n) {
              temp[offset + (depth << 1) - 1] += temp[offset + depth - 1];
            }
          }

          temp[n - 1] = 0;
          for (int depth = (n >> 1); depth >= 1; depth >>= 1) {
            __syncthreads(); 
            int offset = idx * (depth << 1); 
            if (offset < n) {
              int t = temp[offset + depth - 1];
              temp[offset + depth - 1] = temp[offset + (depth << 1) - 1];
              temp[offset + (depth << 1) - 1] += t; 
            }
          }
          __syncthreads(); 
          data[2 * idx] = temp[2 * idx];
          data[2 * idx + 1] = temp[2 * idx + 1];
        }

        __global__ void kernBlockIncrements(int n, int* data, int* sum) {
          int idx = 2 * ((blockIdx.x * blockDim.x) + threadIdx.x); 
          if (idx >= n) {
            return; 
          }
          data[idx] += sum[blockIdx.x];
          data[idx + 1] += sum[blockIdx.x];
        }

        void _scan(int n, int* dev_data) {
          int numBlocks = blocksPerGrid((n >> 1));  // enough blocks that can handle 2 elements per thread, up to n elements
          int numThreads = BLOCKSIZE;

          if (numBlocks == 1) {
            kernEfficientScan<<<numBlocks, numThreads, n * sizeof(int)>>>(n, dev_data);
            checkCUDAError("kernEfficientScan failed");
          }
          else {
            int* dev_sum = nullptr; 
            hipMalloc((void**)&dev_sum, numBlocks * sizeof(int)); 
            checkCUDAError("hipMalloc dev_sum failed"); 

            int numElementsPerBlock = numThreads << 1; 

            kernEfficientScanMultiBlock<<<numBlocks, numThreads, numElementsPerBlock * sizeof(int)>>>(numElementsPerBlock, dev_data, dev_sum); 
            checkCUDAError("kernEfficientScanMultiBlock failed");

            // perform (exclusive) scan
            _scan(numBlocks, dev_sum); 

            // perform sums on dev_data
            kernBlockIncrements<<<numBlocks, numThreads>>>(n, dev_data, dev_sum);
            checkCUDAError("kernBlockIncrements dev_sum failed");

            // inclusive to exclusive scan by shifting the results and inserting identity
            int* dev_temp = nullptr; 
            hipMalloc((void**)&dev_temp, n * sizeof(int)); 
            checkCUDAError("hipMalloc dev_temp failed");

            hipMemset(dev_temp, 0, 1 * sizeof(int)); 
            checkCUDAError("hipMemset dev_temp failed");

            hipMemcpy(dev_temp + 1, dev_data, (n - 1) * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy dev_data to dev_temp faile");

            hipMemcpy(dev_data, dev_temp, n * sizeof(int), hipMemcpyDeviceToDevice); 
            checkCUDAError("hipMemcpy dev_temp to dev_data failed");

            // ??? profit
            hipFree(dev_sum); 
            hipFree(dev_temp); 
          }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int arrSize = n;
            if (n & (n - 1)) {  // if n is not a power of 2, pad the array to next power of 2
              arrSize = 1 << ilog2ceil(n);
            }

            int* dev_data = nullptr; 

            hipMalloc((void**)&dev_data, sizeof(int) * arrSize);
            checkCUDAError("hipMalloc dev_data failed");

            hipMemset(dev_data, 0, sizeof(int) * arrSize); 
            checkCUDAError("hipMalloc dev_data failed");

            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_data failed");

            _scan(arrSize, dev_data); 

            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_data failed");

            hipFree(dev_data);
            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int arrSize = n;
            if (n & (n - 1)) {  // if n is not a power of 2, pad the array to next power of 2
              arrSize = 1 << ilog2ceil(n);
            }

            int numBlocks = blocksPerGrid(arrSize);
            int numThreads = BLOCKSIZE;

            int* dev_idata = nullptr; 
            int* dev_odata = nullptr; 

            hipMalloc((void**)&dev_idata, arrSize * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed");

            hipMalloc((void**)&dev_odata, arrSize * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed");

            hipMemcpy(dev_idata, idata, arrSize * sizeof(int), hipMemcpyHostToDevice); 
            checkCUDAError("hipMemcpy dev_data failed");

            // create boolean array
            int* dev_bool = nullptr; 
            hipMalloc((void**)&dev_bool, arrSize * sizeof(int));
            checkCUDAError("hipMalloc dev_bool failed");

            hipMemset(dev_bool, 0, arrSize * sizeof(int)); 
            checkCUDAError("hipMemset dev_bool failed");

            StreamCompaction::Common::kernMapToBoolean<<<numBlocks, numThreads>>>(n, dev_bool, dev_idata);
            checkCUDAError("kernMapToBoolean failed");

            // create indices array and copy bools data
            int* dev_indices = nullptr;
            hipMalloc((void**)&dev_indices, arrSize * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed");

            hipMemcpy(dev_indices, dev_bool, arrSize * sizeof(int), hipMemcpyDeviceToDevice);
            checkCUDAError("hipMemcpy dev_indices failed");

            // perform scan on boolean array
            _scan(arrSize, dev_indices); 

            // scatter
            StreamCompaction::Common::kernScatter<<<numBlocks, numThreads>>>(n, dev_odata, dev_idata, dev_bool, dev_indices); 
            checkCUDAError("StreamCompaction failed");

            // get the output size
            int odata_size = 0; 
            hipMemcpy(&odata_size, dev_indices + arrSize - 1, 1 * sizeof(int), hipMemcpyDeviceToHost); 
            checkCUDAError("hipMemcpy odata_size failed"); 

            // copy output into host memory
            hipMemcpy(odata, dev_odata, odata_size * sizeof(int), hipMemcpyDeviceToHost); 
            checkCUDAError("hipMemcpy odata failed");

            timer().endGpuTimer();

            hipFree(dev_bool); 
            hipFree(dev_idata); 
            hipFree(dev_odata); 
            hipFree(dev_indices); 
            return odata_size;
        }
    }
}
