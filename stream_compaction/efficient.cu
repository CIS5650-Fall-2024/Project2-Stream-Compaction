#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 128 

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;

        // Initialise device variables to use in compact()
        int *dev_bools, *dev_idata, *dev_odata, *dev_scanResult;

        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // n & (n - 1): This expression clears the lowest set bit of n. 
        // If n is a power of 2, it has exactly one bit set, and subtracting 1 from it flips 
        // all the bits after the most significant bit. Performing n & (n - 1) results in 
        // 0 if and only if n is a power of 2.
        // n > 0: This ensures that n is positive, since negative numbers and zero are not powers of 2.
        __device__ __host__ bool isPowerOf2(int n) {
            return (n > 0) && ((n & (n - 1)) == 0);
        }

        __global__ void upsweep(int n, int d, int *data) {
            int k = threadIdx.x + (blockIdx.x * blockDim.x);

            if (k >= n) {
                return;
            }

            // is that shifting bits to the left by n positions is equivalent to multiplying the number by 2^n.
            // faster than calling pow(2, n)
            int two_pow_d_plus_1 = 1 << (d + 1);

            if (k % two_pow_d_plus_1 != 0) {
                return;
            }

            int two_pow_d = 1 << d;
            data[k + two_pow_d_plus_1 - 1] += data[k + two_pow_d - 1];
        }

        __global__ void init_downsweep(int n, int *odata) {
            int i = threadIdx.x + (blockIdx.x * blockDim.x);

            if (i == n - 1) {
                odata[i] = 0;
            } 
        }

        __global__ void downsweep(int n, int d, int *data) {
            int k = threadIdx.x + (blockIdx.x * blockDim.x);

            if (k >= n) {
                return;
            }

            // is that shifting bits to the left by n positions is equivalent to multiplying the number by 2^n.
            // faster than calling pow(2, n)
            int two_pow_d_plus_1 = 1 << (d + 1);

            if (k % two_pow_d_plus_1 != 0) {
                return;
            }

            int two_pow_d = 1 << d;
            int t = data[k + two_pow_d - 1];
            data[k + two_pow_d - 1] = data[k + two_pow_d_plus_1 - 1];
            data[k + two_pow_d_plus_1 - 1] += t;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            int *dev_odata_local;
            // Your intermediate array sizes will need to be rounded to the next power of two.
            int rounded_n = isPowerOf2(n) ? n : 1 << ilog2ceil(n);
   
            hipMalloc((void**)&dev_odata_local, rounded_n * sizeof(int));

            // Copy idata to dev_odata_local first
            // Although we might initialise dev_odata_local with more than n elements, idata only contains n elements
            hipMemcpy(dev_odata_local, idata, n * sizeof(int), hipMemcpyHostToDevice);

            // Set up the grid and block sizes
            dim3 fullBlocksPerGrid((rounded_n + blockSize - 1) / blockSize);

            // upsweep
            for (int d = 0; d <= ilog2ceil(rounded_n) - 1; d++) {
                upsweep << <fullBlocksPerGrid, blockSize >> > (rounded_n, d, dev_odata_local);
            }

            // downsweep
            init_downsweep << <fullBlocksPerGrid, blockSize >> > (rounded_n, dev_odata_local);

            for (int d = ilog2ceil(rounded_n) - 1; d >= 0; d--) {
                downsweep << <fullBlocksPerGrid, blockSize >> > (rounded_n, d, dev_odata_local);
            }

            // Copy the result back to the host
            // Note that odata is only supposed to have n elements
            hipMemcpy(odata, dev_odata_local, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_odata_local);

            timer().endGpuTimer();
        }

        /************************************************************************************
         * Define another scan function so that it can be called from compact()
         * Here dev_bools has already been initialised in compact()
         ************************************************************************************/
        void scan_without_timer(int n, int *odata, const int *idata) {
            // Initialise dev_scanResult. dev_bools only has n elements.
            hipMemcpy(odata, idata, n * sizeof(int), hipMemcpyDeviceToDevice);

            // Your intermediate array sizes will need to be rounded to the next power of two.
            int rounded_n = isPowerOf2(n) ? n : 1 << ilog2ceil(n);

            // Set up the grid and block sizes
            dim3 fullBlocksPerGrid((rounded_n + blockSize - 1) / blockSize);

            // upsweep
            for (int d = 0; d <= ilog2ceil(rounded_n) - 1; d++) {
                upsweep << <fullBlocksPerGrid, blockSize >> > (rounded_n, d, odata);
            }

            // downsweep
            init_downsweep << <fullBlocksPerGrid, blockSize >> > (rounded_n, odata);

            for (int d = ilog2ceil(rounded_n) - 1; d >= 0; d--) {
                downsweep << <fullBlocksPerGrid, blockSize >> > (rounded_n, d, odata);
            }
        }
 
        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            // Initialise host variables for returning
            int *scanResult = new int[n];
            
            // Allocate memory on the device
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            hipMalloc((void**)&dev_bools, n * sizeof(int));
            hipMalloc((void**)&dev_scanResult, n * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            // Set up the grid and block sizes
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);

            // Map idata to a 0/1 array
            Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (n, dev_bools, dev_idata);

            // Scan the boolean array
            scan_without_timer(n, dev_scanResult, dev_bools); // n will be rounded in the scan function

            // Perform scatter
            Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_scanResult);

            hipMemcpy(scanResult, dev_scanResult, n * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            int count = n == 0 ? 0 : scanResult[n - 1] + (idata[n - 1] != 0 ? 1 : 0);

            // Clean up device memory
            hipFree(dev_idata);
            hipFree(dev_bools);
            hipFree(dev_scanResult);
            hipFree(dev_odata);

            delete[] scanResult;

            timer().endGpuTimer();

            return count;
        }
    }
}
