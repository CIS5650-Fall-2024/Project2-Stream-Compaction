#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int d, int* buffer) {
            int index = threadIdx.x + blockIdx.x * blockDim.x;
   
            int pow2tod = 1 << d;
            int pow2todp1 = 2 * pow2tod;

            if (index > n / pow2todp1 - 1) return;
            index *= pow2todp1;

            buffer[index + pow2todp1 - 1] += buffer[index + pow2tod - 1];
        }

        __global__ void kernDownSweep(int n, int d, int s, int* buffer) {
            int index = threadIdx.x + blockIdx.x * blockDim.x;

            int pow2tod = 1 << d;
            int pow2todp1 = 2 * pow2tod;

            if (s) {
                buffer[pow2todp1 - 1] = 0;
            }

            if (index > n / pow2todp1 - 1) return;
            index *= pow2todp1;

            int tmp = buffer[index + pow2tod - 1];
            buffer[index + pow2tod - 1] = buffer[index + pow2todp1 - 1];
            buffer[index + pow2todp1 - 1] += tmp;
        }

        __global__ void kernZeroPadding(int n, int d, int* buffer) {
            int index = threadIdx.x + blockIdx.x * blockDim.x;

            if (index >= 1 << (d + 1) - n) return;

            buffer[n + index] = 0;
        }

        dim3 computeBlocksPerGrid(int threads, int blockSize) {
            return dim3{ (unsigned int)(threads + blockSize - 1) / blockSize };
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int blockSize = 128;

            bool isPower2Length = (n == (1 << ilog2(n)));

            int bufferLength = (isPower2Length) ? n : 1 << ilog2ceil(n);

            int* tmpArray;
            hipMalloc((void**)&tmpArray, bufferLength * sizeof(int));
            checkCUDAError("hipMalloc tmpArray failed!");

            if (!isPower2Length) {
                dim3 blocks = computeBlocksPerGrid(n - (1 << ilog2(n)), blockSize);
                kernZeroPadding<<<blocks, blockSize>>>(n, ilog2(n), tmpArray);
                checkCUDAError("kernZeroPadding failed!");
                hipDeviceSynchronize();
            }

            hipMemcpy(tmpArray, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            // TODO
            for (int d = 0; d < ilog2ceil(n); ++d) {
                dim3 blocks = computeBlocksPerGrid(bufferLength / (1 << (d + 1)), blockSize);
                kernUpSweep<<<blocks, blockSize>>>(bufferLength, d, tmpArray);
                checkCUDAError("kernUpSweep failed!");
                hipDeviceSynchronize();
            }
            
            bool flag = 1;
            for (int d = ilog2ceil(n) - 1; d >= 0; --d) {
                dim3 blocks = computeBlocksPerGrid(bufferLength / (1 << (d + 1)), blockSize);
                kernDownSweep<<<blocks, blockSize>>>(bufferLength, d, flag, tmpArray);
                flag = 0;
                checkCUDAError("kernDownSweep failed!");
                hipDeviceSynchronize();
            }
            timer().endGpuTimer();

            hipMemcpy(odata, tmpArray, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(tmpArray);
        }

        void scanUntimed(int n, int* odata, const int* idata) {
            int blockSize = 128;

            bool isPower2Length = (n == (1 << ilog2(n)));

            int bufferLength = (isPower2Length) ? n : 1 << ilog2ceil(n);

            int* tmpArray;
            hipMalloc((void**)&tmpArray, bufferLength * sizeof(int));
            checkCUDAError("hipMalloc tmpArray failed!");

            if (!isPower2Length) {
                dim3 blocks = computeBlocksPerGrid(n - (1 << ilog2(n)), blockSize);
                kernZeroPadding << <blocks, blockSize >> > (n, ilog2(n), tmpArray);
                checkCUDAError("kernZeroPadding failed!");
                hipDeviceSynchronize();
            }

            hipMemcpy(tmpArray, idata, n * sizeof(int), hipMemcpyHostToDevice);

            // TODO
            for (int d = 0; d < ilog2ceil(n); ++d) {
                dim3 blocks = computeBlocksPerGrid(bufferLength / (1 << (d + 1)), blockSize);
                kernUpSweep<<<blocks, blockSize>>>(bufferLength, d, tmpArray);
                checkCUDAError("kernUpSweep failed!");
                hipDeviceSynchronize();
            }

            bool flag = 1;
            for (int d = ilog2ceil(n) - 1; d >= 0; --d) {
                dim3 blocks = computeBlocksPerGrid(bufferLength / (1 << (d + 1)), blockSize);
                kernDownSweep<<<blocks, blockSize>>>(bufferLength, d, flag, tmpArray);
                flag = 0;
                checkCUDAError("kernDownSweep failed!");
                hipDeviceSynchronize();
            }

            hipMemcpy(odata, tmpArray, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(tmpArray);
        }


        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            int blockSize = 128;
            dim3 blocks{ (unsigned int)(n + blockSize - 1) / blockSize };

            int* dev_buffer1;
            int* dev_buffer2;
            int* dev_boolArray;
            int* dev_indices;
            hipMalloc((void**)&dev_boolArray, n * sizeof(int));
            checkCUDAError("hipMalloc dev_boolArray failed!");
            hipMalloc((void**)&dev_indices,   n * sizeof(int));
            hipDeviceSynchronize();
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMalloc((void**)&dev_buffer1,   n * sizeof(int));
            checkCUDAError("hipMalloc dev_buffer1 failed!");
            hipMalloc((void**)&dev_buffer2,   n * sizeof(int));
            checkCUDAError("hipMalloc dev_buffer2 failed!");

            hipMemcpy(dev_buffer1, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            hipDeviceSynchronize();
            checkCUDAError("timer failed!");
            // TODO
            

            StreamCompaction::Common::kernMapToBoolean<<<blocks, blockSize>>>(n, dev_boolArray, dev_buffer1);
            hipDeviceSynchronize();
            checkCUDAError("kernMapToBoolean failed!");
            
            scanUntimed(n, dev_indices, dev_boolArray);

            StreamCompaction::Common::kernScatter<<<blocks, blockSize>>>(n, dev_buffer2, dev_buffer1, dev_boolArray, dev_indices);
            checkCUDAError("kernScatter failed!");
            hipDeviceSynchronize();
            
            hipMemcpy(odata, dev_buffer2, n * sizeof(int), hipMemcpyDeviceToHost);
            
            int numElem;
            hipMemcpy(&numElem, &dev_indices[n - 1], sizeof(int), hipMemcpyDeviceToHost);
            
            timer().endGpuTimer();

            hipFree(dev_boolArray);
            hipFree(dev_indices);
            hipFree(dev_buffer1);
            hipFree(dev_buffer2);
            return numElem;
        }
    }
}