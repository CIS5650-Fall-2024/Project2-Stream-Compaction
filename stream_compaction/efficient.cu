#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <>

#define blockSize 1024

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __host__ __device__ int pow_two(int t) {
            return 1 << t;
        }

        __global__ void up_sweep(int n, int d, int* odata, const int* idata) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index < n) {
                odata[index] = idata[index];
                int step = pow_two(d + 1);
                if (index % step == 0) {
                    int t = pow_two(d);
                    odata[index + step - 1] = idata[index + step - 1] + idata[index + t - 1];
                }
            }
        }

        __global__ void down_sweep(int n, int d, int* odata, int* idata) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index < n) {
                int step = pow_two(d + 1);
                odata[index] = idata[index];
                if (index % step == 0) {
                    int t = pow_two(d);
                    odata[index + t - 1] = idata[index + step - 1];
                    odata[index + step - 1] += idata[index + t - 1];
                }
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int size = pow_two(ilog2ceil(n));
            int depth = ilog2ceil(size);
            dim3 fullBlockPerGrid = ((size + blockSize - 1) / blockSize);

            int* buffer1;
            int* buffer2;
            hipMalloc((void**)&buffer1, size * sizeof(int));
            checkCUDAErrorFn("failed to allocate buffer1");

            hipMalloc((void**)&buffer2, size * sizeof(int));
            checkCUDAErrorFn("failed to allocate buffer2");

            hipMemcpy(buffer1, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();

            for (int d = 0; d < depth; d++) {
                up_sweep << <fullBlockPerGrid, blockSize >> > (size, d, buffer2, buffer1);
                std::swap(buffer1, buffer2);
            }

            hipMemset(buffer1 + size - 1, 0, sizeof(int));

            for (int d = depth - 1; d >= 0; d--) {
                down_sweep << <fullBlockPerGrid, blockSize >> > (size, d, buffer2, buffer1);
                std::swap(buffer1, buffer2);
            }

            timer().endGpuTimer();

            hipMemcpy(odata, buffer1, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(buffer1);
            hipFree(buffer2);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO
            int size = pow_two(ilog2ceil(n));
            int depth = ilog2ceil(size);
            dim3 fullBlockPerGrid = ((size + blockSize - 1) / blockSize);

            int* bools;
            int* indices;
            int* buffer1;
            int* buffer2;

            hipMalloc((void**)&bools, size * sizeof(int));
            checkCUDAErrorFn("failed to allocate bools");

            hipMalloc((void**)&indices, size * sizeof(int));
            checkCUDAErrorFn("failed to allocate indices");

            hipMalloc((void**)&buffer1, size * sizeof(int));
            checkCUDAErrorFn("failed to allocate buffer1");

            hipMalloc((void**)&buffer2, size * sizeof(int));
            checkCUDAErrorFn("failed to allocate buffer2");

            hipMemcpy(buffer1, idata, n * sizeof(int), hipMemcpyHostToDevice);

            Common::kernMapToBoolean << <fullBlockPerGrid, blockSize >> > (size, bools, buffer1);

            hipMemcpy(indices, bools, n * sizeof(int), hipMemcpyDeviceToDevice);

            for (int d = 0; d < depth; d++) {
                up_sweep << <fullBlockPerGrid, blockSize >> > (size, d, buffer2, indices);
                std::swap(indices, buffer2);
            }

            hipMemset(indices + size - 1, 0, sizeof(int));

            for (int d = depth - 1; d >= 0; d--) {
                down_sweep << <fullBlockPerGrid, blockSize >> > (size, d, buffer2, indices);
                std::swap(indices, buffer2);
            }

            Common::kernScatter << <fullBlockPerGrid, blockSize >> > (size, buffer2, buffer1, bools, indices);

            int count = 0;
            hipMemcpy(&count, indices + size - 1, sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(odata, buffer2, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(bools);
            hipFree(indices);
            hipFree(buffer1);
            hipFree(buffer2);

            timer().endGpuTimer();
            return count;
        }
    }
}
