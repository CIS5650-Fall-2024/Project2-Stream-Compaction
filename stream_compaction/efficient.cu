#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <vector>

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        // Kernel for the upsweep phase of the scan
        __global__ void kern_upsweep(int* data, int offset, int n) {
            int tid = threadIdx.x + blockIdx.x * blockDim.x;
            int index = tid * offset * 2;

            if (index + offset * 2 - 1 < n) {
                data[index + offset * 2 - 1] += data[index + offset - 1];
            }
        }

        // Kernel for the downsweep phase of the scan
        __global__ void kern_downsweep(int* data, int offset, int n) {
            int tid = threadIdx.x + blockIdx.x * blockDim.x;
            int index = tid * offset * 2;

            if (index + offset * 2 - 1 < n) {
                int temp = data[index + offset - 1];
                data[index + offset - 1] = data[index + offset * 2 - 1];
                data[index + offset * 2 - 1] += temp;
            }
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int rounded_size = pow(2, ilog2ceil(n));

            // Allocate memory for device input/output arrays
            int* dev_data;
            hipMalloc((void**)&dev_data, rounded_size * sizeof(int));
            checkCUDAError("hipMalloc dev_data failed!");

            // Copy input data to device, with padding for non-power-of-two sizes
            hipMemcpy(dev_data, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_data failed");

            if (rounded_size > n) {
                hipMemset(dev_data + n, 0, (rounded_size - n) * sizeof(int));
                checkCUDAError("hipMemset dev_data failed");
            }

            timer().startGpuTimer();

            // Perform the upsweep phase
            for (int offset = 1; offset < rounded_size; offset *= 2) {
                int numBlocks = (rounded_size / (offset * 2) + blockSize - 1) / blockSize;
                kern_upsweep << <numBlocks, blockSize >> > (dev_data, offset, rounded_size);
                checkCUDAError("kern_upsweep failed!");
                hipDeviceSynchronize();
            }

            // Set the last element to 0 (this is required by the downsweep phase)
            hipMemset(dev_data + rounded_size - 1, 0, sizeof(int));
            checkCUDAError("hipMemset dev_data failed");

            // Perform the downsweep phase
            for (int offset = rounded_size / 2; offset >= 1; offset /= 2) {
                int numBlocks = (rounded_size / (offset * 2) + blockSize - 1) / blockSize;
                kern_downsweep << <numBlocks, blockSize >> > (dev_data, offset, rounded_size);
                checkCUDAError("kern_downsweep failed!");
                hipDeviceSynchronize();
            }
            
            timer().endGpuTimer();

            // Copy the result back to host
            hipMemcpy(odata, dev_data, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy odata failed");

            // Free device memory
            hipFree(dev_data);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            size_t paddedSize = (size_t) 1 << ilog2ceil(n);

            int* dev_idata;
            int* dev_odata;
            int* dev_bools;
            int* dev_indices;

            // Allocate memory for device arrays
            checkCUDAError("failed");
            hipMalloc((void**)&dev_idata, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed");
            hipMalloc((void**)&dev_odata, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed");
            hipMalloc((void**)&dev_bools, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_bools failed");
            hipMalloc((void**)&dev_indices, paddedSize * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed");

            // Copy input data to device
            hipMemcpy(dev_idata, idata, sizeof(int) * n, hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy dev_idata failed");

            if (paddedSize > n) {
                hipMemset(dev_idata + n, 0, (paddedSize - n) * sizeof(int));
                checkCUDAError("hipMemset dev_data failed");
            }

            dim3 fullBlocksPerGrid((paddedSize + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            
            // Step 1: Map to Boolean
            StreamCompaction::Common::kernMapToBoolean << <fullBlocksPerGrid, blockSize >> > (paddedSize, dev_bools, dev_idata);
            checkCUDAError("kernMapToBoolean failed");

            // Step 2: Perform Scan on Boolean Array
            hipMemcpy(dev_indices, dev_bools, sizeof(int) * paddedSize, hipMemcpyDeviceToDevice);

            // Up-sweep phase
            for (int offset = 1; offset < paddedSize; offset *= 2) {
                int numBlocks = (paddedSize / (offset * 2) + blockSize - 1) / blockSize;
                if (numBlocks > 0) { // Only run if there is work to do
                    kern_upsweep << <numBlocks, blockSize >> > (dev_indices, offset, paddedSize);
                    checkCUDAError("kern_upsweep failed!");
                    hipDeviceSynchronize();
                }
            }

            // Set the last element to 0 (this is required by the downsweep phase)
            hipMemset(dev_indices + paddedSize - 1, 0, sizeof(int));
            checkCUDAError("hipMemset failed");

            // Down-sweep phase
            for (int offset = 1; offset < paddedSize; offset *= 2) {
                int numBlocks = (paddedSize / (offset * 2) + blockSize - 1) / blockSize;
                if (numBlocks > 0) { // Only run if there is work to do
                    kern_downsweep << <numBlocks, blockSize >> > (dev_indices, offset, paddedSize);
                    checkCUDAError("kern_downsweep failed");
                    hipDeviceSynchronize();
                }
            }

            // Step 3: Scatter
            StreamCompaction::Common::kernScatter << <fullBlocksPerGrid, blockSize >> > (paddedSize, dev_odata, dev_idata, dev_bools, dev_indices);
            checkCUDAError("kernScatter failed");

            timer().endGpuTimer();

            // Step 4: Copy results and free memory
            std::vector<int> a;
            a.resize(paddedSize);
            hipMemcpy(a.data(), dev_indices, paddedSize * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("test failed");



            int compactedSize;
            hipMemcpy(&compactedSize, dev_indices + paddedSize - 1, sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy compactedSize failed");
            hipMemcpy(odata, dev_odata, sizeof(int) * compactedSize, hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy odata failed");

            // Free memory
            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_bools);
            hipFree(dev_indices);


            return compactedSize;
        }
    }
}
