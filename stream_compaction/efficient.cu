#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include <iostream>

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernEfficientScan(int n, int* data) {
          int idx = threadIdx.x;
          if (idx >= (n >> 1)) {
            return; 
          }
          extern __shared__ int temp[]; 
          temp[2 * idx]     = data[2 * idx];
          temp[2 * idx + 1] = data[2 * idx + 1];
          
          // up sweep 
          for (int depth = 1; depth < n; depth <<= 1) {
            __syncthreads();
            int offset = idx * (depth << 1);  // k * 2^(d+1)
            if (offset < n) {
              temp[offset + (depth << 1) - 1] += temp[offset + depth - 1];
            }
          }

          temp[n - 1] = 0;
          for (int depth = (n >> 1); depth >= 1; depth >>= 1) {
            __syncthreads(); 
            int offset = idx * (depth << 1); 
            if (offset < n) {
              int t = temp[offset + depth - 1];
              temp[offset + depth - 1] = temp[offset + (depth << 1) - 1];
              temp[offset + (depth << 1) - 1] += t; 
            }
          }
          __syncthreads(); 
          data[2 * idx] = temp[2 * idx];
          data[2 * idx + 1] = temp[2 * idx + 1];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();

            int* dev_data = nullptr; 

            hipMalloc((void**)&dev_data, sizeof(int) * n); 
            hipMemcpy(dev_data, idata, sizeof(int) * n, hipMemcpyHostToDevice); 

            int numBlocks = blocksPerGrid((n >> 1));
            int numThreads = BLOCKSIZE; 

            // call kernel 
            kernEfficientScan<<<numBlocks, numThreads, n>>>(n, dev_data);

            hipMemcpy(odata, dev_data, sizeof(int) * n, hipMemcpyDeviceToHost); 
            hipFree(dev_data); 

            hipDeviceSynchronize(); 
            timer().endGpuTimer();
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            
            // todo
            timer().endGpuTimer();
            return -1;
        }
    }
}
