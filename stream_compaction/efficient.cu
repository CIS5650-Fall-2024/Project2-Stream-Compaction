#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"
#include ""

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernUpSweep(int n, int pow2tod, int* buffer) {
            int index = threadIdx.x + blockIdx.x * blockDim.x;
   
            int pow2todp1 = 2 * pow2tod;

            if (index > n / pow2todp1 - 1) return;
            index *= pow2todp1;

            buffer[index + pow2todp1 - 1] += buffer[index + pow2tod - 1];
        }

        __global__ void kernDownSweep(int n, int pow2tod, int* buffer) {
            int index = threadIdx.x + blockIdx.x * blockDim.x;

            int pow2todp1 = 2 * pow2tod;

            if (index > n / pow2todp1 - 1) return;
            index *= pow2todp1;

            int tmp = buffer[index + pow2tod - 1];
            buffer[index + pow2tod - 1] = buffer[index + pow2todp1 - 1];
            buffer[index + pow2todp1 - 1] += tmp;
        }

        dim3 computeBlocksPerGrid(int threads, int blockSize) {
            return dim3{ (unsigned int)(threads + blockSize - 1) / blockSize };
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata, bool timed) {
            int blockSize = 128;

            bool isPower2Length = (n == (1 << ilog2(n)));

            int bufferLength = (isPower2Length) ? n : 1 << ilog2ceil(n);

            int* dev_tmpArray;
            hipMalloc((void**)&dev_tmpArray, bufferLength * sizeof(int));
            checkCUDAError("hipMalloc tmpArray failed!");

            if (!isPower2Length) {
                hipMemset(dev_tmpArray + n, 0, (bufferLength - n) * sizeof(int));
            }

            hipMemcpy(dev_tmpArray, idata, n * sizeof(int), hipMemcpyHostToDevice);

            if (timed) timer().startGpuTimer();
            // TODO
            for (int d = 0; d < ilog2ceil(n); ++d) {
                dim3 blocks = computeBlocksPerGrid(bufferLength / (1 << (d + 1)), blockSize);
                kernUpSweep<<<blocks, blockSize>>>(bufferLength, 1 << d, dev_tmpArray);
                hipDeviceSynchronize();
                checkCUDAError("kernUpSweep failed!");
            }

            hipMemset(dev_tmpArray + bufferLength - 1, 0, sizeof(int));
            
            for (int d = ilog2ceil(n) - 1; d >= 0; --d) {
                dim3 blocks = computeBlocksPerGrid(bufferLength / (1 << (d + 1)), blockSize);
                kernDownSweep<<<blocks, blockSize>>>(bufferLength, 1 << d, dev_tmpArray);
                hipDeviceSynchronize();
                checkCUDAError("kernDownSweep failed!");
            }
            if (timed) timer().endGpuTimer();

            hipMemcpy(odata, dev_tmpArray, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_tmpArray);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            int blockSize = 128;
            dim3 blocks{ (unsigned int)(n + blockSize - 1) / blockSize };

            int* dev_buffer1;
            int* dev_buffer2;
            int* dev_boolArray;
            int* dev_indices;
            hipMalloc((void**)&dev_boolArray, n * sizeof(int));
            checkCUDAError("hipMalloc dev_boolArray failed!");
            hipMalloc((void**)&dev_indices,   n * sizeof(int));
            checkCUDAError("hipMalloc dev_indices failed!");
            hipMalloc((void**)&dev_buffer1,   n * sizeof(int));
            checkCUDAError("hipMalloc dev_buffer1 failed!");
            hipMalloc((void**)&dev_buffer2,   n * sizeof(int));
            checkCUDAError("hipMalloc dev_buffer2 failed!");

            hipMemcpy(dev_buffer1, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy idata->dev_buffer1 failed!");

            timer().startGpuTimer();

            StreamCompaction::Common::kernMapToBoolean<<<blocks, blockSize>>>(n, dev_boolArray, dev_buffer1);
            hipDeviceSynchronize();
            checkCUDAError("kernMapToBoolean failed!");
            
            scan(n, dev_indices, dev_boolArray, 0);

            StreamCompaction::Common::kernScatter<<<blocks, blockSize>>>(n, dev_buffer2, dev_buffer1, dev_boolArray, dev_indices);
            hipDeviceSynchronize();
            checkCUDAError("kernScatter failed!");
            
            hipMemcpy(odata, dev_buffer2, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy dev_buffer2->odata failed!");
            
            // Index that last element in idata would have, if it was valid
            int lastIndex;
            hipMemcpy(&lastIndex, dev_indices + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            // Check if last element is valid
            int lastBool;
            hipMemcpy(&lastBool, dev_boolArray + n - 1, sizeof(int), hipMemcpyDeviceToHost);
            
            timer().endGpuTimer();

            hipFree(dev_boolArray);
            hipFree(dev_indices);
            hipFree(dev_buffer1);
            hipFree(dev_buffer2);
            return (lastBool) ? lastIndex + 1 : lastIndex;
        }
    }
}