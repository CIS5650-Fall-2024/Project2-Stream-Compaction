#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define blockSize 512

namespace StreamCompaction {
	namespace Efficient {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

		__global__ void kernUpSweep(int n, int d, int* odata)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) return;

			int k = index * (1 << (d + 1));

			odata[k + (1 << (d + 1)) - 1] += odata[k + (1 << d) - 1];
		}

		__global__ void kernDownSweep(int n, int d, int* odata)
		{
			int index = (blockIdx.x * blockDim.x) + threadIdx.x;
			if (index >= n) return;

			int k = index * (1 << (d + 1));

			int t = odata[k + (1 << d) - 1];
			odata[k + (1 << d) - 1] = odata[k + (1 << (d + 1)) - 1];
			odata[k + (1 << (d + 1)) - 1] += t;
		}

		/**
		 * Performs prefix-sum (aka scan) on idata, storing the result into odata.
		 */
		void scan(int n, int* odata, const int* idata) {
			timer().startGpuTimer();

			int* dev_indices;
			hipMalloc((void**)&dev_indices, n * sizeof(int));
			hipMemcpy(dev_indices, idata, n * sizeof(int), hipMemcpyHostToDevice);

			//upsweep
			dim3 blocksPerGrid((n + blockSize - 1) / blockSize);
			for (int d = 0; d < ilog2ceil(n); d++)
			{
				int kSteps = n >> (d + 1);
				dim3 blocksPerGrid((kSteps + blockSize - 1) / blockSize);
				kernUpSweep << < blocksPerGrid, blockSize >> > (kSteps, d, dev_indices);
			}

			//downsweep
			hipMemcpy(odata, dev_indices, n * sizeof(int), hipMemcpyDeviceToHost);
			odata[n - 1] = 0;
			hipMemcpy(dev_indices, odata, n * sizeof(int), hipMemcpyHostToDevice);

			for (int d = ilog2ceil(n) - 1; d >= 0; d--)
			{
				int kSteps = n >> (d + 1);
				// non-power-of-2
				if (kSteps << (d + 1) != n)  kSteps = n >> d;
				dim3 blocksPerGrid((kSteps + blockSize - 1) / blockSize);
				kernDownSweep << < blocksPerGrid, blockSize >> > (kSteps, d, dev_indices);
			}

			hipMemcpy(odata, dev_indices, n * sizeof(int), hipMemcpyDeviceToHost);
			hipFree(dev_indices);
			timer().endGpuTimer();
		}

		/**
		 * Performs stream compaction on idata, storing the result into odata.
		 * All zeroes are discarded.
		 *
		 * @param n      The number of elements in idata.
		 * @param odata  The array into which to store elements.
		 * @param idata  The array of elements to compact.
		 * @returns      The number of elements remaining after compaction.
		 */
		int compact(int n, int* odata, const int* idata) {
			timer().startGpuTimer();
			//a buffer used to r/w single element of device ptr
			int* buffer = new int[n];

			int* dev_idata;
			int* dev_label;
			hipMalloc((void**)&dev_idata, n * sizeof(int));
			hipMalloc((void**)&dev_label, n * sizeof(int));
			hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

			//get labels
			dim3 blocksPerGrid((n + blockSize - 1) / blockSize);
			Common::kernMapToBoolean << <blocksPerGrid, blockSize >> > (n, dev_label, dev_idata);

			//scan begin
			int* dev_indices;
			hipMalloc((void**)&dev_indices, n * sizeof(int));
			hipMemcpy(dev_indices, dev_label, n * sizeof(int), hipMemcpyDeviceToDevice);

			//upsweep
			for (int d = 0; d < ilog2ceil(n); d++)
			{
				int kSteps = n >> (d + 1);
				dim3 blocksPerGrid((kSteps + blockSize - 1) / blockSize);
				kernUpSweep << < blocksPerGrid, blockSize >> > (kSteps, d, dev_indices);
			}

			//downsweep
			hipMemcpy(buffer, dev_indices, n * sizeof(int), hipMemcpyDeviceToHost);
			buffer[n - 1] = 0;
			hipMemcpy(dev_indices, buffer, n * sizeof(int), hipMemcpyHostToDevice);

			for (int d = ilog2ceil(n) - 1; d >= 0; d--)
			{
				int kSteps = n >> (d + 1);
				// non-power-of-2
				if (kSteps << (d + 1) != n)  kSteps = n >> d;
				dim3 blocksPerGrid((kSteps + blockSize - 1) / blockSize);
				kernDownSweep << < blocksPerGrid, blockSize >> > (kSteps, d, dev_indices);
			}

			//calculate total numbers
			hipMemcpy(buffer, dev_indices, n * sizeof(int), hipMemcpyDeviceToHost);
			int count = buffer[n - 1];
			hipMemcpy(buffer, dev_label, n * sizeof(int), hipMemcpyDeviceToHost);
			count += buffer[n - 1];

			//scatter to labeled data
			int* dev_odata;
			hipMalloc((void**)&dev_odata, count * sizeof(int));
			Common::kernScatter << <blocksPerGrid, blockSize >> > (n, dev_odata,
				dev_idata, dev_label, dev_indices);

			//copy to cpu output
			hipMemcpy(odata, dev_odata, count * sizeof(int), hipMemcpyDeviceToHost);

			delete[] buffer;
			hipFree(dev_idata);
			hipFree(dev_odata);
			hipFree(dev_label);
			hipFree(dev_indices);
			timer().endGpuTimer();
			return count;
		}
	}
}
