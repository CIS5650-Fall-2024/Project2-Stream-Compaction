#include "hip/hip_runtime.h"
#include "common.h"

void checkCUDAErrorFn(const char *msg, const char *file, int line) {
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
}


namespace StreamCompaction {
    namespace Common {

        /**
         * Maps an array to an array of 0s and 1s for stream compaction. Elements
         * which map to 0 will be removed, and elements which map to 1 will be kept.
         */
        __global__ void kernMapToBoolean(int n, int *bools, const int *idata) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) return;

            bools[index] = (idata[index] == 0);
        }

        /**
         * Performs scatter on an array. That is, for each element in idata,
         * if bools[idx] == 1, it copies idata[idx] to odata[indices[idx]].
         */
        __global__ void kernScatter(int n, int *odata,
                const int *idata, const int *scannedBools) {
            int threadId = threadIdx.x + (blockIdx.x * blockDim.x);
            if (threadId >= n) return;

            int data = idata[threadId];
            int scan_i = scannedBools[threadId];

            // Special case for last element of idata array
            if (threadId == n - 1 && data) {
                odata[scan_i] = data;
                return;
            }

            int scan_iplusone = scannedBools[threadId + 1];
            if (scan_i != scan_iplusone) {
                odata[scan_i] = data;
            }
        }

    }
}
