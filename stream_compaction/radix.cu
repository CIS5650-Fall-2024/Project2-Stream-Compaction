#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "radix.h"

#include "common.h"
#include "efficient.h"

#define blockSize 128

namespace StreamCompaction {
    namespace Radix {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernMapToCurrBits(int n, int *odata, const int *idata, int bit) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            // get the bit-th bit of the number, but also flip the bit to produce the e array
            odata[index] = !((idata[index] >> bit) & 1);
        }

        __global__ void kernComputeT(int n, int *odata, const int *idata, const int totalFalses) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            odata[index] = index - idata[index] + totalFalses;
        }

        __global__ void kernComputeD(int n, int *odata, const int *be, const int *t, const int *f) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            // be array is stored as e array. To access b, reverse the condition
            odata[index] = be[index] ? f[index] : t[index];
        }

        __global__ void scatter(int n, int *odata, const int *idata, const int *d) {
            int index = threadIdx.x + (blockIdx.x * blockDim.x);
            if (index >= n) {
                return;
            }

            odata[d[index]] = idata[index];
        }

        int getMaxBits(int n, const int *idata) {
            // find maximum number in the array
            int maxNum = 0;
            for (int i = 0; i < n; i++) {
                maxNum = std::max(maxNum, idata[i]);
            }
            // calculate the number of bits of the maximum number
            int maxBits = 0;
            while (maxNum > 0) {
                maxNum >>= 1;
                maxBits++;
            }
            return maxBits;
        }

        /**
         * Performs radix sort on idata, storing the result into odata.
         * @param n the number of elements in idata
         * @param odata output.txt data
         * @param idata input data
         * @param maxBits the maximum number of bits of a given number in the array
         */
        void sort(int n, int *odata, const int *idata, int maxBits) {
            int* dev_idata;
            int* dev_odata;
            int* dev_be;
            int* dev_f;
            int* dev_t;
            int* dev_d;
            int fullBlocksPerGrid = (n + blockSize - 1) / blockSize;

            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed!");
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed!");
            hipMalloc((void**)&dev_be, n * sizeof(int));
            checkCUDAError("hipMalloc dev_b failed!");
            hipMalloc((void**)&dev_f, n * sizeof(int));
            checkCUDAError("hipMalloc dev_f failed!");
            hipMalloc((void**)&dev_t, n * sizeof(int));
            checkCUDAError("hipMalloc dev_t failed!");
            hipMalloc((void**)&dev_d, n * sizeof(int));
            checkCUDAError("hipMalloc dev_d failed!");

            // copy the input to GPU (size n data)
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            for (int i = 0; i < maxBits; i++) {
                // b & e array (b can be acquired by flipping back)
                kernMapToCurrBits<<<fullBlocksPerGrid, blockSize>>>(n, dev_be, dev_idata, i);
                // exclusive scan f array
                StreamCompaction::Efficient::scan(n, dev_f, dev_be);
                // calculate totalFalses
                int lastFVal = 0;
                int lastEVal = 0;
                hipMemcpy(&lastEVal, dev_be + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&lastFVal, dev_f + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                int totalFalses = lastFVal + lastEVal;
                // t array
                kernComputeT<<<fullBlocksPerGrid, blockSize>>>(n, dev_t, dev_f, totalFalses);
                // d array
                kernComputeD<<<fullBlocksPerGrid, blockSize>>>(n, dev_d, dev_be, dev_t, dev_f);
                // scatter
                scatter<<<fullBlocksPerGrid, blockSize>>>(n, dev_odata, dev_idata, dev_d);
                // swap
                std::swap(dev_idata, dev_odata);
            }
            timer().endGpuTimer();

            // copy the result to odata (size n data)
            hipMemcpy(odata, dev_idata, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_be);
            hipFree(dev_f);
            hipFree(dev_t);
            hipFree(dev_d);
        }
    }
}