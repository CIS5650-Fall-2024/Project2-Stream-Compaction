#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "common.h"
#include "radix.h"

#define blockSize 256

void printArrayT(int n, int* a, bool abridged = false) {
    printf("    [ ");
    for (int i = 0; i < n; i++) {
        if (abridged && i + 2 == 15 && n > 16) {
            i = n - 2;
            printf("... ");
        }
        printf("%3d ", a[i]);
    }
    printf("]\n");
}
namespace StreamCompaction {
    namespace Radix {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void kernBitMaskNot(int n, int digit, int* odata, int* idata) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            int bitmask = 1 << digit;

            odata[idx] = (int)(!(bool)(idata[idx] & bitmask));
        }

        __global__ void kernScatter(int n, int fCount, int* odata_scat, int* f, int* e, int* idata) {
            // odata = d array, idata = i array
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            int actualIdx = !e[idx] ? idx - f[idx] + fCount : f[idx];
            odata_scat[actualIdx] = idata[idx];
        }

        // copied from naive.cu
        __global__ void kernScan(int n, int depth, int* odata, int* idata) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            if (idx < depth) {
                odata[idx] = idata[idx];
                return;
            }

            odata[idx] = idata[idx - depth] + idata[idx];
            return;
        }

        __global__ void kernToExclusive(int n, int* odata, int* idata) {
            int idx = blockIdx.x * blockDim.x + threadIdx.x;
            if (idx >= n) return;

            if (idx == 0) {
                odata[idx] = 0;
            }
            else {
                odata[idx] = idata[idx - 1];
            }
            return;
        }

        /**
         * Performs radixSort on idata, storing the result into odata.
         */
        void radixSort(int n, int *odata, const int *idata) {
            int* dev_odata;
            int* dev_idata;

            int* dev_odata_scat;
            int* dev_idata_const;

            int* dev_temp;

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed");
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed");

            hipMalloc((void**)&dev_odata_scat, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata_const failed");
            hipMalloc((void**)&dev_idata_const, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata_const failed");

            hipMalloc((void**)&dev_temp, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata_const failed");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to dev_idata failed");
            hipMemcpy(dev_idata_const, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to dev_idata_const failed");

            int* test = new int[n];

            // find the maximum number (to set the number of iterations of bitmask)
            int maxDigitLen = ilog2ceil(n);

            if (time)
                timer().startGpuTimer();
            // TODO

            // upsweep
            for (int d = 0; d < maxDigitLen; d++) {
                dim3 blocksPerGrid((n + blockSize - 1) / blockSize);

                // produce 'e' array to dev_odata
                kernBitMaskNot<<<blocksPerGrid, blockSize>>>(n, d, dev_odata, dev_idata_const);

                int fCount, tempCount;
                hipMemcpy(&tempCount, dev_odata + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                std::swap(dev_idata, dev_odata); // store 'e' to dev_idata

                // exclusive scan to get 'f' array to dev_odata (copied from naive.cu)
                kernScan<<<blocksPerGrid, blockSize >> > (n, 1, dev_temp, dev_idata);
                for (int depth = 2; depth <= ilog2ceil(n); ++depth) {
                    kernScan<<<blocksPerGrid, blockSize>>>(n, 1 << (depth - 1), dev_odata, dev_temp);
                    std::swap(dev_odata, dev_temp);
                }
                kernToExclusive<<<blocksPerGrid, blockSize>>>(n, dev_odata, dev_temp);

                hipMemcpy(&fCount, dev_odata + n - 1, sizeof(int), hipMemcpyDeviceToHost);
                fCount += tempCount;

                // scatter to output array from 'f', 'e', and 'i' arrays respectively
                kernScatter<<<blocksPerGrid, blockSize>>>(n, fCount, dev_odata_scat, dev_odata, dev_idata, dev_idata_const);

                // pass scatter result to next iteration
                std::swap(dev_odata_scat, dev_idata_const);
            }

            if (time)
                timer().endGpuTimer();

            std::swap(dev_odata_scat, dev_idata_const);

            hipMemcpy(odata, dev_odata_scat, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy from dev_odata_scat failed");

            hipFree(dev_odata);
            checkCUDAError("hipFree dev_odata failed");
            hipFree(dev_idata);
            checkCUDAError("hipFree dev_idata failed");
            hipFree(dev_odata_scat);
            checkCUDAError("hipFree dev_odata_scat failed");
            hipFree(dev_idata_const);
            checkCUDAError("hipFree dev_idata_const failed");
            hipFree(dev_temp);
            checkCUDAError("hipFree dev_temp failed");
        }
    }
}
