#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_odata;
            int* dev_idata;

            hipMalloc((void**)&dev_odata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_odata failed");
            hipMalloc((void**)&dev_idata, n * sizeof(int));
            checkCUDAError("hipMalloc dev_idata failed");

            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAError("hipMemcpy to dev_idata failed");

            thrust::device_ptr<int> thrust_dev_idata(dev_idata);
            thrust::device_ptr<int> thrust_dev_odata(dev_odata);

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());    
            
            thrust::exclusive_scan(thrust_dev_idata, thrust_dev_idata +n, thrust_dev_odata);

            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAError("hipMemcpy from dev_odata failed");

            hipFree(dev_odata);
            checkCUDAError("hipFree dev_odata failed");
            hipFree(dev_idata);
            checkCUDAError("hipFree dev_idata failed");
        }
    }
}
