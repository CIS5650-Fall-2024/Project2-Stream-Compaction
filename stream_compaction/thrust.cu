#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:

            thrust::device_vector<int> dev_thrust_in(idata, idata + n);
            thrust::device_vector<int> dev_thrust_out(n);

            thrust::exclusive_scan(dev_thrust_in.begin(), dev_thrust_in.end(), dev_thrust_out.begin());

            timer().endGpuTimer();

            thrust::copy(dev_thrust_out.begin(), dev_thrust_out.end(), odata);
        }
    }
}
