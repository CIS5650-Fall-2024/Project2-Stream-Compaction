#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int* odata, const int* idata) {

            // create thrust device vectors
            thrust::device_vector<int> d_idata(idata, idata + n);
            thrust::device_vector<int> d_odata(n);
            timer().startGpuTimer();

            // use thrust exclusive scan
            thrust::exclusive_scan(d_idata.begin(), d_idata.end(), d_odata.begin());

            timer().endGpuTimer();

            // copy result back
            thrust::copy(d_odata.begin(), d_odata.end(), odata);
        }
    }
}
