#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_buffer;
            thrust::device_ptr<int> dev_thrustBuffer;
            hipMalloc((void**)&dev_buffer, n * sizeof(int));
            checkCUDAError("hipMalloc dev_buffer failed!");
            dev_thrustBuffer = thrust::device_ptr<int>(dev_buffer);

            hipMemcpy(dev_buffer, idata, n * sizeof(int), hipMemcpyHostToDevice);

            timer().startGpuTimer();
            thrust::exclusive_scan(dev_thrustBuffer, dev_thrustBuffer + n, dev_thrustBuffer);
            timer().endGpuTimer();

            hipMemcpy(odata, dev_buffer, n * sizeof(int), hipMemcpyDeviceToHost);

            hipFree(dev_buffer);
        }
    }
}
