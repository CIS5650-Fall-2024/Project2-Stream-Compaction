#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            thrust::host_vector<int>h_vector_in(idata, idata + n);
            thrust::host_vector<int>h_vector_out(odata, odata + n);

            thrust::device_vector<int> d_vector_in = h_vector_in;
            thrust::device_vector<int> d_vector_out = h_vector_out;

            timer().startGpuTimer();
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

            thrust::exclusive_scan(d_vector_in.begin(), d_vector_in.end(), d_vector_out.begin());


            timer().endGpuTimer();
            thrust::copy(d_vector_out.begin(), d_vector_out.end(), odata);
        }
    }
}
