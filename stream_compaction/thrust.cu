#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include "common.h"
#include "thrust.h"

namespace StreamCompaction {
    namespace Thrust {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            // TODO use `thrust::exclusive_scan`
            // example: for device_vectors dv_in and dv_out:
            // thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());

			thrust::host_vector<int> host_in(idata, idata + n);
            thrust::host_vector<int> host_out(odata, odata + n);

			// Cast to device vectors
            thrust::device_vector<int> dev_in = host_in;
            thrust::device_vector<int> dev_out = host_out;

			// Start Timer
			timer().startGpuTimer();
			thrust::exclusive_scan(dev_in.begin(), dev_in.end(), dev_out.begin());
			// End Timer
			timer().endGpuTimer();

            hipMemcpy(odata, dev_out.data().get(), sizeof(int) * n, hipMemcpyDeviceToHost);
        }
    }
}
